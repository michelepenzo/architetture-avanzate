#include "hip/hip_runtime.h"
#include "transposers.hh"

void transposers::serial_csr2csc(
    int m, int n, int nnz, 
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal) {
    
    int* curr = new int[n](); // array inizializzato con tutti '0'

    DPRINT_MSG("Reference")
    DPRINT_ARR(csrRowPtr, m+1)
    DPRINT_ARR(csrColIdx, nnz)
    DPRINT_ARR(csrVal, nnz)

    // 1. costruisco `cscColPtr` come istogramma delle frequenze degli elementi per ogni colonna
    for(int i = 0; i < m; i++) {
        for(int j = csrRowPtr[i]; j < csrRowPtr[i+1]; j++) {
            cscColPtr[csrColIdx[j]+1]++;
        }
    }
    // 2. applico prefix_sum per costruire corretto `cscColPtr` (ogni cella tiene conto dei precedenti)
    for(int i = 1; i < n+1; i++) {
        cscColPtr[i] += cscColPtr[i-1];
    }
    // 3. sistemo indici di riga e valori
    for(int i = 0; i < m; i++) {
        for(int j = csrRowPtr[i]; j < csrRowPtr[i+1]; j++) {
            int col = csrColIdx[j];
            int loc = cscColPtr[col] + curr[col];
            curr[col]++;
            cscRowIdx[loc] = i;
            cscVal[loc] = csrVal[j];
        }
    }

    DPRINT_MSG("SERIAL")
    DPRINT_ARR(cscColPtr, n+1)
    DPRINT_ARR(cscRowIdx, nnz)
    DPRINT_ARR(   cscVal, nnz)

    delete[] curr;
}

void transposers::cuda_wrapper(
    int m, int n, int nnz,
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal,
    algo _algo
) {

    int * csrRowPtr_cuda = utils::cuda::allocate_send<int>(csrRowPtr, m+1);
    int * csrColIdx_cuda = utils::cuda::allocate_send<int>(csrColIdx, nnz);
    float * csrVal_cuda  = utils::cuda::allocate_send<float>(csrVal, nnz);

    int * cscColPtr_cuda = utils::cuda::allocate_zero<int>(n+1);
    int * cscRowIdx_cuda = utils::cuda::allocate_zero<int>(nnz);
    float * cscVal_cuda  = utils::cuda::allocate_zero<float>(nnz);

    _algo(
        m, n, nnz,
        csrRowPtr_cuda, csrColIdx_cuda, csrVal_cuda, 
        cscColPtr_cuda, cscRowIdx_cuda, cscVal_cuda
    );

    utils::cuda::deallocate(csrRowPtr_cuda);
    utils::cuda::deallocate(csrColIdx_cuda);
    utils::cuda::deallocate(csrVal_cuda);

    utils::cuda::deallocate_recv<int>(cscColPtr, cscColPtr_cuda, n+1);
    utils::cuda::deallocate_recv<int>(cscRowIdx, cscRowIdx_cuda, nnz);
    utils::cuda::deallocate_recv<float>(cscVal, cscVal_cuda, nnz);
}

__global__ 
void reorder_elements_kernel(
    int n, int nnz,
    int *inter, int *intra, 
    int *csrRowIdx, int *csrColIdx, float *csrVal,
    int *cscColPtr, int *cscRowIdx, float *cscVal
) {

    const int j = blockIdx.x;
    
    // allineo inter alla riga corretta
    inter += j*n;

    // recupero gli estremi della porzione di array da processare
    const int BLOCK_SIZE = DIV_THEN_CEIL(nnz, HISTOGRAM_BLOCKS);
    const int start = j*BLOCK_SIZE;
    const int end = min((j+1)*BLOCK_SIZE, nnz);
    const int len = end - start;

    // printf("(%2d): START=%d, LEN=%d\n", j, start, len);

    // calcolo la posizione degli elementi
    for(int i = 0; i < len; i++) {
        int cid = csrColIdx[start + i];
        int loc = cscColPtr[cid] + inter[cid] + intra[start + i];
        cscRowIdx[loc] = csrRowIdx[start + i];
        cscVal[loc] = csrVal[start + i];

        // printf("(%2d): i=%2d, start+i=%2d, { cid=%2d | colptr=%2d, inter=%d, intra=%2d }-> loc=%2d, csrVal[start+i]=%2.0f, cscVal[loc]=%2.0f\n",
        //     j, i, start+i, cid, cscColPtr[cid], inter[cid], intra[i], loc, csrVal[start+i], cscVal[loc]);
    }
}

void transposers::scan_csr2csc(
    int m, int n, int nnz,
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal) {

    // 1. espandi l'array di puntatori agli indici di riga, nell'array di indici esteso
    int * csrRowIdx = utils::cuda::allocate_zero<int>(nnz);
    procedures::cuda::pointers_to_indexes(csrRowPtr, m, csrRowIdx, nnz);
    DPRINT_ARR_CUDA(csrRowIdx, nnz)

    // 2. riempi inter, intra, e colPtr
    int * inter;
    int * intra = utils::cuda::allocate_zero<int>(nnz);
    int * colPtr = utils::cuda::allocate_zero<int>(n+1);
    procedures::cuda::indexes_to_pointers(csrColIdx, nnz, &inter, intra, colPtr, n);
    for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
        DPRINT_ARR_CUDA(inter+i*n, n)
    }
    DPRINT_ARR_CUDA(intra, nnz)

    // 3. applica scan ai puntatori
    procedures::cuda::scan(colPtr, cscColPtr, n+1);

    // 4. permuta valori
    reorder_elements_kernel<<<HISTOGRAM_BLOCKS, 1>>>(
        n, nnz,
        inter, intra, 
        csrRowIdx, csrColIdx, csrVal, 
        cscColPtr, cscRowIdx, cscVal);
    CUDA_CHECK_ERROR

    DPRINT_MSG("SCAN")
    DPRINT_ARR_CUDA(cscColPtr, n+1)
    DPRINT_ARR_CUDA(cscRowIdx, nnz)
    DPRINT_ARR_CUDA(cscVal, nnz)
    for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
        DPRINT_ARR_CUDA(inter+i*n, n)
    }
    DPRINT_ARR_CUDA(intra, nnz)

    utils::cuda::deallocate(csrRowIdx);
    utils::cuda::deallocate(inter);
    utils::cuda::deallocate(intra);
    utils::cuda::deallocate(colPtr);
}

__global__ 
void copy_histo_kernel(
    int * colPtrIn,  int * rowIdxIn,  float * valIn,
    int * colPtrOut, int * rowIdxOut, float * valOut,
    int n, int nnz, int BLOCK_SIZE
) {
    int i = blockIdx.x;

    int * colPtrA = colPtrIn + 2*i*(n+1);
    int * colPtrB = colPtrIn + (2*i+1)*(n+1);
    int * colPtrC = colPtrOut + 2*i*(n+1);

    // sistema i puntatori al blocco di uscita
    for(int i = 0; i < n+1; i++) {
        colPtrC[i] = colPtrA[i] + colPtrB[i];
    }

    int * rowIdxA = rowIdxIn + 2*i*BLOCK_SIZE;
    int * rowIdxB = rowIdxIn + (2*i+1)*BLOCK_SIZE;
    int * rowIdxC = rowIdxOut + 2*i*BLOCK_SIZE;

    float * valA = valIn + 2*i*BLOCK_SIZE;
    float * valB = valIn + (2*i+1)*BLOCK_SIZE;
    float * valC = valOut + 2*i*BLOCK_SIZE;

    // copia in output i valori corretti di rowIdx e val
    for(int i = 0; i < n; i++) {

        int sa = colPtrA[i], la = colPtrA[i+1] - sa;
        int sb = colPtrB[i], lb = colPtrB[i+1] - sb;
        int sc = colPtrC[i], lc = colPtrC[i+1] - sc;

        //DPRINT_MSG("Col=%d, sa=%d, sb=%d, sc=%d, la=%d, lb=%d, lc=%d", i, sa, sb, sc, la, lb, lc)

        utils::cuda::devcopy<int>(rowIdxC+sc, rowIdxA+sa, la);
        utils::cuda::devcopy<float>( valC+sc,    valA+sa, la);

        sc = sc + la;
        utils::cuda::devcopy<int>(rowIdxC+sc, rowIdxB+sb, lb);
        utils::cuda::devcopy<float>( valC+sc,    valB+sb, lb);
    }
}

void transposers::merge_csr2csc(
    int m, int n, int nnz, 
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal) {

    // alloca lo spazio necessario per effettuare il sort
    struct merge_buffer {
        //int * colPtrBlock;
        int * colIdx;
        int * rowIdx;
        float * val;
    };
    merge_buffer buffer[2];
    buffer[0].colIdx = utils::cuda::allocate_zero<int>(nnz);
    buffer[0].rowIdx = utils::cuda::allocate_zero<int>(nnz);
    buffer[0].val    = utils::cuda::allocate_zero<float>(nnz);
    buffer[1].colIdx = utils::cuda::allocate_zero<int>(nnz);
    buffer[1].rowIdx = cscRowIdx;
    buffer[1].val    = cscVal;
    int * colPtr     = utils::cuda::allocate_zero<int>(n+1);

    // 1. espandi rowPtr in rowIdx
    DPRINT_MSG("1 ---- row IDX to PTR")
    procedures::cuda::pointers_to_indexes(csrRowPtr, m, buffer[0].rowIdx, nnz);
    DPRINT_ARR_CUDA(buffer[0].rowIdx, nnz);

    // 2. ordina per indice delle colonne
    DPRINT_MSG("2 ---- sort by column index")
    utils::cuda::copy(buffer[0].colIdx, csrColIdx, nnz);
    procedures::cuda::segsort3(buffer[0].colIdx, buffer[1].colIdx, nnz, buffer[0].rowIdx, buffer[1].rowIdx, csrVal, buffer[1].val);
    DPRINT_ARR_CUDA(buffer[1].colIdx,  nnz);
    DPRINT_ARR_CUDA(buffer[1].rowIdx, nnz);
    DPRINT_ARR_CUDA(buffer[1].val,    nnz);

    // 3. i blocchi devono raggiungere dimensione TARGET_BLOCK_SIZE
    DPRINT_MSG("3 ---- merging")
    int full = 1;
    int CURRENT_BLOCK_SIZE = SEGSORT_ELEMENTS_PER_BLOCK;
#ifdef ALTERNATIVE_MERGETRANS
    int TARGET_BLOCK_SIZE = DIV_THEN_CEIL(nnz, HISTOGRAM_BLOCKS);
#else
    int TARGET_BLOCK_SIZE = (nnz-1)*2;
#endif
    DPRINT_MSG("INIT Block Size %d", CURRENT_BLOCK_SIZE)
    DPRINT_ARR_CUDA(buffer[full].colIdx,  nnz);
    DPRINT_ARR_CUDA(buffer[full].rowIdx, nnz);
    DPRINT_ARR_CUDA(buffer[full].val,    nnz);

    while(CURRENT_BLOCK_SIZE < TARGET_BLOCK_SIZE) {
    
        procedures::cuda::segmerge3_sm_step(
            buffer[full].colIdx, buffer[1-full].colIdx, 
            nnz, CURRENT_BLOCK_SIZE, 
            buffer[full].rowIdx, buffer[1-full].rowIdx, 
            buffer[full].val, buffer[1-full].val
        );
    
        full = 1 - full;
        CURRENT_BLOCK_SIZE *= 2;
        DPRINT_MSG("Block size %d", CURRENT_BLOCK_SIZE)
        DPRINT_ARR_CUDA(buffer[full].colIdx,  nnz);
        DPRINT_ARR_CUDA(buffer[full].rowIdx, nnz);
        DPRINT_ARR_CUDA(buffer[full].val,    nnz);
    }

#ifdef ALTERNATIVE_MERGETRANS
    // 4. applico index_to_pointers parallelo
    DPRINT_MSG("Applying index_to_pointers with BLOCK_SIZE=%d", CURRENT_BLOCK_SIZE)
    int * inter;
    procedures::cuda::pre_indexes_to_pointers(buffer[full].colIdx, nnz, &inter, n, CURRENT_BLOCK_SIZE);
    // debug prints...
    for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
        DPRINT_ARR_CUDA(inter+i*(n+1), n+1);
    }

    DPRINT_MSG("Applying scan with BLOCK_SIZE=%d", CURRENT_BLOCK_SIZE)
    int * inter_out = utils::cuda::allocate_zero<int>((HISTOGRAM_BLOCKS+1) * (n+1));
    for(int i = 1; i <= HISTOGRAM_BLOCKS; i++) {
        const int OFFSET = i * (n+1);
        procedures::cuda::scan(inter + OFFSET, inter_out + OFFSET, n+2);
    }
    // debug prints...
    for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
        DPRINT_ARR_CUDA(inter_out+i*(n+1), n+1);
    }

    // 5. merging blocks
    int * interBuffer[2];
    interBuffer[full] = inter_out;
    interBuffer[1-full] = inter;

    DPRINT_MSG("Starting with blocks %d of size %d", HISTOGRAM_BLOCKS, CURRENT_BLOCK_SIZE)
    for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
        DPRINT_ARR_CUDA(interBuffer[full]+i*(n+1), n+1);
    }
    DPRINT_ARR_CUDA(buffer[full].rowIdx, nnz)
    DPRINT_ARR_CUDA(buffer[full].val, nnz)

    for(int blocks = HISTOGRAM_BLOCKS; blocks > 1; blocks /= 2) {

        copy_histo_kernel<<<blocks/2, 1>>>(
            interBuffer[full],   buffer[full].rowIdx,   buffer[full].val,
            interBuffer[1-full], buffer[1-full].rowIdx, buffer[1-full].val,
            n, nnz, CURRENT_BLOCK_SIZE
        );

        // blocks è sempre multiplo di due tranne quando termino

        full = 1 - full;
        CURRENT_BLOCK_SIZE *= 2;

        DPRINT_MSG("After 'copy_histo_kernel' blocks are %d of size %d", blocks, CURRENT_BLOCK_SIZE)
        for(int i = 0; i <= HISTOGRAM_BLOCKS; i++) {
            DPRINT_ARR_CUDA(interBuffer[full]+i*(n+1), n+1);
        }
        DPRINT_ARR_CUDA(buffer[full].rowIdx, nnz)
        DPRINT_ARR_CUDA(buffer[full].val, nnz)
    }

    // 6. copia indietro i risultati
    utils::cuda::copy<int>(interBuffer[full], cscColPtr, n);

    utils::cuda::deallocate(inter);
    utils::cuda::deallocate(inter_out);
#endif
    if(full != 1) {
        utils::cuda::copy<int>(cscRowIdx, buffer[full].rowIdx, nnz);
        utils::cuda::copy<float>(cscVal, buffer[full].val, nnz);
    }

    DPRINT_ARR_CUDA(cscColPtr, n+1);
    DPRINT_ARR_CUDA(cscRowIdx, nnz);
    DPRINT_ARR_CUDA(cscVal,    nnz);

    utils::cuda::deallocate(colPtr);
    utils::cuda::deallocate(buffer[0].colIdx);
    utils::cuda::deallocate(buffer[1].colIdx);
    utils::cuda::deallocate(buffer[0].rowIdx);
    utils::cuda::deallocate(buffer[0].val);
}

const char* _cusparseGetErrorName(int status) {
    switch(status) {
        case HIPSPARSE_STATUS_SUCCESS            : return "HIPSPARSE_STATUS_SUCCESS: the operation completed successfully.";
        case HIPSPARSE_STATUS_NOT_INITIALIZED    : return "HIPSPARSE_STATUS_NOT_INITIALIZED: the library was not initialized.";
        case HIPSPARSE_STATUS_ALLOC_FAILED       : return "HIPSPARSE_STATUS_ALLOC_FAILED: the reduction buffer could not be allocated.";
        case HIPSPARSE_STATUS_INVALID_VALUE      : return "HIPSPARSE_STATUS_INVALID_VALUE: the idxBase is neither HIPSPARSE_INDEX_BASE_ZERO nor HIPSPARSE_INDEX_BASE_ONE.";
        case HIPSPARSE_STATUS_ARCH_MISMATCH      : return "HIPSPARSE_STATUS_ARCH_MISMATCH: the device does not support double precision.";
        case HIPSPARSE_STATUS_EXECUTION_FAILED   : return "HIPSPARSE_STATUS_EXECUTION_FAILED: the function failed to launch on the GPU.";
        case HIPSPARSE_STATUS_INTERNAL_ERROR     : return "HIPSPARSE_STATUS_INTERNAL_ERROR: an internal operation failed (check if you are compiling correctly wrt your GPU architecture).";
        default                                 : return "UNKNOWN ERROR";
    }
}

void cusparse_generic_csr2csc_gpumemory(
    int m, int n, int nnz, 
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal, bool use_algo1) {

    #if (CUDART_VERSION >= 9000) && (CUDART_VERSION < 10000)

        hipsparseHandle_t handle;
        hipsparseStatus_t status;

        // 1. allocate resources
        status = hipsparseCreate(&handle);
        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseCreate: " << _cusparseGetErrorName(status) << std::endl;
            return;
        }

        // 2. call transpose
        // reference: https://docs.nvidia.com/cuda/archive/9.0/cusparse/index.html#cusparse-lt-t-gt-csr2csc
        status = hipsparseScsr2csc(
            handle, 
            m, n, nnz, csrVal, csrRowPtr, csrColIdx, cscVal, cscRowIdx, cscColPtr,
            HIPSPARSE_ACTION_NUMERIC,    // [copyValues] the operation is performed on data and indices.
            HIPSPARSE_INDEX_BASE_ZERO);  // [idxBase]

        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseScsr2csc: " << _cusparseGetErrorName(status) << std::endl;
            hipsparseDestroy(handle);
            return;
        }

        status = hipsparseDestroy(handle);
        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseDestroy: " << _cusparseGetErrorName(status) << std::endl;
            return;
        }

        return;

    #elif (CUDART_VERSION >= 10000) 

        hipsparseHandle_t handle;
        hipsparseStatus_t status;
        size_t buffer_size;

        // 1. allocate resources
        status = hipsparseCreate(&handle);
        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseCreate: " << hipsparseGetErrorName(status) << std::endl;
            return;
        }

        // 2. ask cusparse how much space it needs to operate
        status = hipsparseCsr2cscEx2_bufferSize(
            handle,                     // link to cusparse engine
            m, n, nnz, csrVal, csrRowPtr, csrColIdx, cscVal, cscColPtr, cscRowIdx, 
            HIP_R_32F,                 // [valType] data type of csrVal, cscVal arrays is 32-bit real (non-complex) single precision floating-point
            HIPSPARSE_ACTION_NUMERIC,    // [copyValues] the operation is performed on data and indices.
            HIPSPARSE_INDEX_BASE_ZERO,   // [idxBase]
            (use_algo1 ? HIPSPARSE_CSR2CSC_ALG1 : HIPSPARSE_CSR2CSC_ALG2),
                                        // which algorithm is used? HIPSPARSE_CSR2CSC_ALG1 or HIPSPARSE_CSR2CSC_ALG2
            &buffer_size);              // fill buffer_size variable

        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseCsr2cscEx2_bufferSize: " << hipsparseGetErrorName(status) << std::endl;
            hipsparseDestroy(handle);
            return;
        } else if(buffer_size <= 0) {
            std::cerr << "csr2csc_cusparse - warning: buffer_size is not positive" << std::endl;
        }

        // 3. callocate buffer space
        void* buffer = NULL;
        CUDA_SAFE_CALL( hipMalloc(&buffer, buffer_size) );
        //std::cout << "Needed " << buffer_size << " bytes to esecute Csr2csc" << std::endl; 

        // 4. call transpose
        status = hipsparseCsr2cscEx2(
            handle, 
            m, n, nnz, csrVal, csrRowPtr, csrColIdx, cscVal, cscColPtr, cscRowIdx, 
            HIP_R_32F,                 // [valType] data type of csrVal, cscVal arrays is 32-bit real (non-complex) single precision floating-point
            HIPSPARSE_ACTION_NUMERIC,    // [copyValues] the operation is performed on data and indices.
            HIPSPARSE_INDEX_BASE_ZERO,   // [idxBase]
            (use_algo1 ? HIPSPARSE_CSR2CSC_ALG1 : HIPSPARSE_CSR2CSC_ALG2),
                                        // which algorithm is used? HIPSPARSE_CSR2CSC_ALG1 or HIPSPARSE_CSR2CSC_ALG2
            buffer);                    // cuda buffer

        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseCsr2cscEx2: " << hipsparseGetErrorName(status) << std::endl;
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseCsr2cscEx2: " << hipsparseGetErrorString(status) << std::endl;
            CUDA_SAFE_CALL( hipFree( buffer ) );
            hipsparseDestroy(handle);
            return;
        }

        CUDA_SAFE_CALL( hipFree( buffer ) );
        status = hipsparseDestroy(handle);
        if(status != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "csr2csc_cusparse - Error while calling hipsparseDestroy: " << hipsparseGetErrorName(status) << std::endl;
            return;
        }
        
    #else

        #error "Versione di CUDA non supportata"

    #endif
}

void transposers::cusparse1_csr2csc(
    int m, int n, int nnz, 
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal)
{
    cusparse_generic_csr2csc_gpumemory(
        m, n, nnz, 
        csrRowPtr, csrColIdx, csrVal, 
        cscColPtr, cscRowIdx, cscVal, 
        true
    );
}

void transposers::cusparse2_csr2csc(
    int m, int n, int nnz, 
    int* csrRowPtr, int* csrColIdx, float* csrVal, 
    int* cscColPtr, int* cscRowIdx, float* cscVal)
{
    cusparse_generic_csr2csc_gpumemory(
        m, n, nnz, 
        csrRowPtr, csrColIdx, csrVal, 
        cscColPtr, cscRowIdx, cscVal, 
        false
    );
}