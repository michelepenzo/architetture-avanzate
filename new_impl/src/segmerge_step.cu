#include "hip/hip_runtime.h"
#include "procedures.hh"

__global__
void segmerge_step_kernel(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {

    int couple_block_id = blockIdx.x;
    int start_1 = 2 * couple_block_id * BLOCK_SIZE;
    int start_2 = (2 * couple_block_id + 1) * BLOCK_SIZE;
    int end_1 = min((2 * couple_block_id + 1) * BLOCK_SIZE, len);
    int end_2 = min((2 * couple_block_id + 2) * BLOCK_SIZE, len);

    int current_1 = start_1;
    int current_2 = start_2;
    int current_output = start_1;
    
    // merge
    while(current_1 < end_1 && current_2 < end_2) {
        if(input[current_1] <= input[current_2]) {
            output[current_output] = input[current_1];
            current_1++;
        } else {
            output[current_output] = input[current_2];
            current_2++;
        }
        current_output++;
    }

    // finisco le rimanenze del primo blocco
    utils::cuda::devcopy<int>(output + current_output, input + current_1, end_1 - current_1);

    // finisco le rimanenze del secondo blocco
    utils::cuda::devcopy<int>(output + current_output, input + current_2, end_2 - current_2);
}

NUMERIC_TEMPLATE(T)
__global__
void segmerge3_step_kernel(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, T INPUT_ARRAY b_in, T * b_out) {

    int couple_block_id = blockIdx.x;
    int start_1 = 2 * couple_block_id * BLOCK_SIZE;
    int start_2 = min((2 * couple_block_id + 1) * BLOCK_SIZE, len);
    int end_1 = min((2 * couple_block_id + 1) * BLOCK_SIZE, len);
    int end_2 = min((2 * couple_block_id + 2) * BLOCK_SIZE, len);

    int current_1 = start_1;
    int current_2 = start_2;
    int current_output = start_1;
    
    // merge
    while(current_1 < end_1 && current_2 < end_2) {
        if(input[current_1] <= input[current_2]) {
            output[current_output] = input[current_1];
            a_out[current_output]  = a_in[current_1];
            b_out[current_output]  = b_in[current_1];
            current_1++;
        } else {
            output[current_output] = input[current_2];
            a_out[current_output]  = a_in[current_2];
            b_out[current_output]  = b_in[current_2];
            current_2++;
        }
        current_output++;
    }

    // finisco le rimanenze del primo blocco
    utils::cuda::devcopy<int>(output + current_output, input + current_1, end_1 - current_1);
    utils::cuda::devcopy<int>(a_out  + current_output, a_in  + current_1, end_1 - current_1);
    utils::cuda::devcopy<T>(  b_out  + current_output, b_in  + current_1, end_1 - current_1);

    // finisco le rimanenze del secondo blocco
    utils::cuda::devcopy<int>(output + current_output, input + current_2, end_2 - current_2);
    utils::cuda::devcopy<int>(a_out  + current_output, a_in  + current_2, end_2 - current_2);
    utils::cuda::devcopy<T>(  b_out  + current_output, b_in  + current_2, end_2 - current_2);
}

void procedures::cuda::segmerge_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {

    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    segmerge_step_kernel<<<DIV_THEN_CEIL(BLOCK_NUMBER, 2), 1>>>(input, output, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR
}

void procedures::cuda::segmerge3_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, int INPUT_ARRAY b_in, int * b_out) {

    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    segmerge3_step_kernel<int><<<DIV_THEN_CEIL(BLOCK_NUMBER, 2), 1>>>(input, output, len, BLOCK_SIZE, a_in, a_out, b_in, b_out);
    CUDA_CHECK_ERROR
}

void procedures::cuda::segmerge3_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, float INPUT_ARRAY b_in, float * b_out) {

    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    segmerge3_step_kernel<float><<<DIV_THEN_CEIL(BLOCK_NUMBER, 2), 1>>>(input, output, len, BLOCK_SIZE, a_in, a_out, b_in, b_out);
    CUDA_CHECK_ERROR
}

void procedures::reference::segmerge_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {
    segmerge3_step(input, output, len, BLOCK_SIZE, NULL, NULL, NULL, NULL);
}

void procedures::reference::segmerge3_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, float INPUT_ARRAY b_in, float * b_out) {

    bool all_three = a_in != NULL && a_out != NULL && b_in != NULL && b_out != NULL;
    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);

    for(int couple_block_id = 0; couple_block_id < DIV_THEN_CEIL(BLOCK_NUMBER, 2); couple_block_id++) {

        DPRINT_MSG("Processing couple_block_id %d\n", couple_block_id)

        int start_1 = 2 * couple_block_id * BLOCK_SIZE;
        int start_2 = min((2 * couple_block_id + 1) * BLOCK_SIZE, len);
        int end_1 = min((2 * couple_block_id + 1) * BLOCK_SIZE, len);
        int end_2 = min((2 * couple_block_id + 2) * BLOCK_SIZE, len);

        DPRINT_MSG("A[%d:%d] B[%d:%d]\n", start_1, end_1, start_2, end_2)

        int current_1 = start_1;
        int current_2 = start_2;
        int current_output = start_1;
        
        // merge
        while(current_1 < end_1 && current_2 < end_2) {
            if(input[current_1] <= input[current_2]) {
                output[current_output] = input[current_1];
                if(all_three) a_out[current_output]  = a_in[current_1];
                if(all_three) b_out[current_output]  = b_in[current_1];
                current_1++;
            } else {
                output[current_output] = input[current_2];
                if(all_three) a_out[current_output]  = a_in[current_2];
                if(all_three) b_out[current_output]  = b_in[current_2];
                current_2++;
            }
            current_output++;
        }

        // finisco le rimanenze del primo blocco
        utils::copy_array<int>(output + current_output, input + current_1, end_1 - current_1);
        if(all_three) utils::copy_array<int>(a_out + current_output, a_in + current_1, end_1 - current_1);
        if(all_three) utils::copy_array<float>(b_out + current_output, b_in + current_1, end_1 - current_1);

        // finisco le rimanenze del secondo blocco
        utils::copy_array<int>(output + current_output, input + current_2, end_2 - current_2);
        if(all_three) utils::copy_array<int>(a_out + current_output, a_in + current_2, end_2 - current_2);
        if(all_three) utils::copy_array<float>(b_out + current_output, b_in + current_2, end_2 - current_2);
    }

}
