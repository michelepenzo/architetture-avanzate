#include "hip/hip_runtime.h"
#include "transposer.hh"

// ===============================================================================
// INDEX TO POINTERS =============================================================
// ===============================================================================

__global__ 
void histogram_blocks_kernel(int INPUT_ARRAY elements, int n_elem, int * histogram_blocks, int hist_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id
    
    if(i < n_elem) {
        int j = blockIdx.x % HISTOGRAM_BLOCKS;
        // each of the HISTOGRAM_BLOCKS grid works on a different partial histogram
        int * this_histogram_block = histogram_blocks + j * hist_len;
        // needs to be atomic: two thread may (unlikely) access the same element at the same time
        // [alternative: one single thread per block (inefficient?)]
        atomicAdd(this_histogram_block + elements[i] + 1, 1);
    }
}

__global__ 
void histogram_merge_kernel(int INPUT_ARRAY histogram_blocks, int hist_len, int * histogram) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id

    if(i < hist_len) {
        int sum = 0;
        for(int k = 0; k < HISTOGRAM_BLOCKS; k++) {
            int * this_histogram_block = histogram_blocks + k * hist_len;
            sum += this_histogram_block[i];
        }
        histogram[i] = sum;
    }
}

void transposer::cuda::indexes_to_pointers(int INPUT_ARRAY idx, int idx_len, int * ptr, int ptr_len) {
    
    DPRINT_MSG("Begin allocation of array with len=%d", HISTOGRAM_BLOCKS * ptr_len)
    int* histogram_blocks = utils::cuda::allocate_zero<int>(HISTOGRAM_BLOCKS * ptr_len);

    DPRINT_MSG("Calling 'histogram_blocks_kernel' with grid=%d, blocks=%d", DIV_THEN_CEIL(idx_len, 1024), 1024)
    histogram_blocks_kernel<<<DIV_THEN_CEIL(idx_len, 1024), 1024>>>(idx, idx_len, histogram_blocks, ptr_len);
    DPRINT_ARR_CUDA(histogram_blocks, HISTOGRAM_BLOCKS * ptr_len)
    CUDA_CHECK_ERROR

    DPRINT_MSG("Calling 'histogram_merge_kernel' with grid=%d, blocks=%d, allocated shared=%d", ptr_len, 1, HISTOGRAM_BLOCKS)
    histogram_merge_kernel<<<DIV_THEN_CEIL(ptr_len, 1024), 1024>>>(histogram_blocks, ptr_len, ptr);
    CUDA_CHECK_ERROR

    utils::cuda::deallocate(histogram_blocks);
}

void transposer::reference::indexes_to_pointers(int INPUT_ARRAY idx, int idx_len, int * ptr, int ptr_len) {
    for(int i = 0; i < idx_len; i++) {
        ASSERT_LIMIT(idx[i]+1, ptr_len);
        ptr[idx[i]+1]++;
    }
}

bool transposer::component_test::indexes_to_pointers() {

    const int N = 10000, NNZ = 10000;
    // input
    int *idx = utils::random::generate_array(0, N-1, NNZ);
    DPRINT_ARR(idx, NNZ)

    // reference implementation
    int *ptr = new int[N+1];
    transposer::reference::indexes_to_pointers(idx, NNZ, ptr, N+1);
    DPRINT_ARR(ptr, N+1)

    // cuda implementation
    int *idx_cuda = utils::cuda::allocate_send<int>(idx, NNZ);
    int *ptr_cuda = utils::cuda::allocate_zero<int>(N+1);
    transposer::cuda::indexes_to_pointers(idx_cuda, NNZ, ptr_cuda, N+1);
    int *ptr2 = new int[N+1]; utils::cuda::recv(ptr2, ptr_cuda, N+1);
    DPRINT_ARR(ptr2, N+1)

    bool ok = utils::equals<int>(ptr, ptr2, N+1);

    utils::cuda::deallocate(idx_cuda);
    utils::cuda::deallocate(ptr_cuda);
    delete idx, ptr, ptr2;
    
    return ok;
} 

// ===============================================================================
// POINTERS TO INDEX =============================================================
// ===============================================================================

__global__ 
void pointers_to_indexes_kernel(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id
    if(i < ptr_len) {
        int start = ptr[i], end = ptr[i+1];
        for(int j = start; j < end; j++) {
            idx[j] = i;
        }
    }
}

void transposer::cuda::pointers_to_indexes(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    pointers_to_indexes_kernel<<<ptr_len, 1>>>(ptr, ptr_len, idx, idx_len);
    CUDA_CHECK_ERROR
}

void transposer::reference::pointers_to_indexes(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    for(int j = 0; j < ptr_len; j++) {
        for(int i = ptr[j]; i < ptr[j+1]; i++) {
            ASSERT_LIMIT(i, idx_len);
            idx[i] = j;
        }
    }
}

bool transposer::component_test::pointers_to_indexes() {

    const int N = 10000, NNZ = 10000;

    int *ptr = utils::random::generate_array(0, 1, N+1);
    ptr[N] = 0;
    utils::prefix_sum(ptr, N+1);
    DPRINT_ARR(ptr, N+1)

    // reference implementation
    int *idx = new int[NNZ];
    reference::pointers_to_indexes(ptr, N+1, idx, NNZ);
    DPRINT_ARR(idx, NNZ)

    // cuda implementation
    int *ptr_cuda = utils::cuda::allocate_send<int>(ptr, N+1);
    int *idx_cuda = utils::cuda::allocate_zero<int>(NNZ);
    transposer::cuda::pointers_to_indexes(ptr_cuda, N+1, idx_cuda, NNZ);
    int *idx2 = new int[N+1]; utils::cuda::recv(idx2, idx_cuda, NNZ);
    DPRINT_ARR(idx2, NNZ)

    bool ok = utils::equals<int>(idx, idx2, NNZ);

    utils::cuda::deallocate(idx_cuda);
    utils::cuda::deallocate(ptr_cuda);
    delete ptr, idx, idx2;
    
    return ok;
}
