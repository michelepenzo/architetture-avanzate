#include "hip/hip_runtime.h"
#include "transposer.hh"

matrix::SparseMatrix* transposer::transpose(matrix::SparseMatrix *sm, Mode mode) {

    matrix::SparseMatrix* result = NULL;
    int esito = COMPUTATION_ERROR;

    if(mode == SERIAL) {
        result = new matrix::SparseMatrix(sm->n, sm->m, sm->nnz, matrix::ALL_ZEROS_INITIALIZATION);
        esito = reference::serial_csr2csc(
            sm->m, sm->n, sm->nnz, 
            sm->csrRowPtr, sm->csrColIdx, sm->csrVal,
            result->csrRowPtr, result->csrColIdx, result->csrVal);
        
    } else if(mode == MERGE) {
        // result = new matrix::SparseMatrix(sm->n, sm->m, sm->nnz, matrix::ALL_ZEROS_INITIALIZATION);
        // esito = merge_host_csr2csc(
        //     11, sm->m, sm->n, sm->nnz, 
        //     sm->csrRowPtr, sm->csrColIdx, sm->csrVal,
        //     result->csrRowPtr, result->csrColIdx, result->csrVal);
        // 
    } else {
        return NULL;
    }

    if(esito == COMPUTATION_ERROR) {
        if(result != NULL) { delete result; }
        return NULL;
    } else {
        return result;
    } 
}

// ===============================================================================
// INDEX TO POINTERS =============================================================
// ===============================================================================

#define HISTOGRAM_BLOCKS 2

__global__ 
void histogram_blocks_kernel(int INPUT_ARRAY elements, int n_elem, int * histogram_blocks, int hist_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id
    
    if(i < n_elem) {
        int j = blockIdx.x % HISTOGRAM_BLOCKS;
        // each of the HISTOGRAM_BLOCKS grid works on a different partial histogram
        int * this_histogram_block = histogram_blocks + j * hist_len;
        // needs to be atomic: two thread may (unlikely) access the same element at the same time
        // [alternative: one single thread per block (inefficient?)]
        atomicAdd(this_histogram_block + elements[i] + 1, 1);
    }
}

__global__ 
void histogram_merge_kernel(int INPUT_ARRAY histogram_blocks, int hist_len, int * histogram) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id

    if(i < hist_len) {
        int sum = 0;
        for(int k = 0; k < HISTOGRAM_BLOCKS; k++) {
            int * this_histogram_block = histogram_blocks + k * hist_len;
            sum += this_histogram_block[i];
        }
        histogram[i] = sum;
    }
}

void transposer::cuda::indexes_to_pointers(int INPUT_ARRAY idx, int idx_len, int * ptr, int ptr_len) {
    
    DPRINT_MSG("Begin allocation of array with len=%d", HISTOGRAM_BLOCKS * ptr_len)
    int* histogram_blocks = utils::cuda::allocate_zero<int>(HISTOGRAM_BLOCKS * ptr_len);

    DPRINT_MSG("Calling 'histogram_blocks_kernel' with grid=%d, blocks=%d", DIV_THEN_CEIL(idx_len, 1024), 1024)
    histogram_blocks_kernel<<<DIV_THEN_CEIL(idx_len, 1024), 1024>>>(idx, idx_len, histogram_blocks, ptr_len);
    DPRINT_ARR_CUDA(histogram_blocks, HISTOGRAM_BLOCKS * ptr_len)
    CUDA_CHECK_ERROR

    DPRINT_MSG("Calling 'histogram_merge_kernel' with grid=%d, blocks=%d, allocated shared=%d", ptr_len, 1, HISTOGRAM_BLOCKS)
    histogram_merge_kernel<<<DIV_THEN_CEIL(ptr_len, 1024), 1024>>>(histogram_blocks, ptr_len, ptr);
    CUDA_CHECK_ERROR

    utils::cuda::deallocate(histogram_blocks);
}

void transposer::reference::indexes_to_pointers(int INPUT_ARRAY idx, int idx_len, int * ptr, int ptr_len) {
    for(int i = 0; i < idx_len; i++) {
        ASSERT_LIMIT(idx[i]+1, ptr_len);
        ptr[idx[i]+1]++;
    }
}

// ===============================================================================
// POINTERS TO INDEX =============================================================
// ===============================================================================

__global__ 
void pointers_to_indexes_kernel(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // global_id
    if(i < ptr_len) {
        int start = ptr[i], end = ptr[i+1];
        for(int j = start; j < end; j++) {
            idx[j] = i;
        }
    }
}

void transposer::cuda::pointers_to_indexes(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    pointers_to_indexes_kernel<<<ptr_len, 1>>>(ptr, ptr_len, idx, idx_len);
    CUDA_CHECK_ERROR
}

void transposer::reference::pointers_to_indexes(int INPUT_ARRAY ptr, int ptr_len, int * idx, int idx_len) {
    for(int j = 0; j < ptr_len; j++) {
        for(int i = ptr[j]; i < ptr[j+1]; i++) {
            ASSERT_LIMIT(i, idx_len);
            idx[i] = j;
        }
    }
}

// ===============================================================================
// SCAN ==========================================================================
// ===============================================================================

#define SCAN_THREAD_PER_BLOCK 512
#define SCAN_ELEMENTS_PER_BLOCK (2*SCAN_THREAD_PER_BLOCK)


__global__ 
void add_kernel(int INPUT_ARRAY array, int INPUT_ARRAY incs, int len) {

	int i = blockIdx.x * SCAN_ELEMENTS_PER_BLOCK + threadIdx.x;
	int b = blockIdx.x;
	if(i < len) {
        array[i] += incs[b];
    }
}

__global__
void scan_kernel(int INPUT_ARRAY input, int * output, int len, int * sums) {

    extern __shared__ int temp[]; // TODO allocazione FISSA ha incremento performace?

    int blockID = blockIdx.x;
	int blockOffset = blockID * SCAN_ELEMENTS_PER_BLOCK;
    int i = threadIdx.x;

    // caricamento dei dati in shared memory: ogni thread carica esattamente due elementi
    temp[2*i]   = (blockOffset + 2*i   < len) ? input[blockOffset + 2*i]   : 0;
    temp[2*i+1] = (blockOffset + 2*i+1 < len) ? input[blockOffset + 2*i+1] : 0;

    // Blelloch Scan
    int offset = 1;

    // prima parte dell'algoritmo: ogni elemento viene sommato
    // al successivo, poi a quello x2 in avanti, poi x4, ..., xD con D=log_2(powtwo)
    for(int d = SCAN_ELEMENTS_PER_BLOCK/2; d > 0; d = d/2) {
        __syncthreads();
        if(i < d) {
            int ai = offset * (2*i + 1) - 1;
			int bi = offset * (2*i + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
    }
    __syncthreads();

    // se sono il thread 0 allora metto a zero l'ultimo elemento
    if(i == 0) {
        sums[blockID] = temp[SCAN_ELEMENTS_PER_BLOCK - 1];
        temp[SCAN_ELEMENTS_PER_BLOCK-1] = 0;
    }

    // seconda parte dell'algoritmo: "downsweep"
    for (int d = 1; d < SCAN_ELEMENTS_PER_BLOCK; d *= 2) // traverse down tree & build scan
	{
		offset /= 2;
		__syncthreads();

		if (i < d)
		{
			int ai = offset * (2*i + 1) - 1;
			int bi = offset * (2*i + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

    __syncthreads();
    // scrivo in output i risultati
    if(2*i < len) { 
        output[blockOffset + 2*i] = temp[2*i];
    }
    if(2*i+1 < len) { 
        output[blockOffset + 2*i+1] = temp[2*i+1];
    }
}

void scan_small(int INPUT_ARRAY input, int * output, int len) {

    // TODO testare diff performance settando SUMS == null
    int *sums = utils::cuda::allocate_zero<int>(1);

    scan_kernel<<< 1, SCAN_THREAD_PER_BLOCK, 2 * SCAN_ELEMENTS_PER_BLOCK * sizeof(int) >>>(
        input, output, len, sums);
    CUDA_CHECK_ERROR

    utils::cuda::deallocate<int>(sums);
}

void scan_large(int INPUT_ARRAY input, int * output, int len) {
    
    const int BLOCKS = DIV_THEN_CEIL(len, SCAN_ELEMENTS_PER_BLOCK);
    int *sums = utils::cuda::allocate_zero<int>(BLOCKS);
    int *incs = utils::cuda::allocate_zero<int>(BLOCKS);
    
    // 1. chiamo il kernel
    scan_kernel<<< BLOCKS, SCAN_THREAD_PER_BLOCK, 2 * SCAN_ELEMENTS_PER_BLOCK * sizeof(int) >>>(
        input, output, len, sums);
    CUDA_CHECK_ERROR

    // 2. ricorsivamente applico scan a sums per ottenere l'array di incrementi
    transposer::cuda::scan(sums, incs, BLOCKS);

    // 3. ad ogni cella del blocco 'i' aggiungo l'incremento 'incs[i]'
    add_kernel<<< BLOCKS, SCAN_ELEMENTS_PER_BLOCK >>>(output, incs, len);
    CUDA_CHECK_ERROR

    utils::cuda::deallocate<int>(sums);
    utils::cuda::deallocate<int>(incs);
}


void transposer::cuda::scan(int INPUT_ARRAY input, int * output, int len) {

    if(len <= SCAN_ELEMENTS_PER_BLOCK) {
        // scan senza array di somme temporaneo
        scan_small(input, output, len);
    } else {
        // scan con array somme temporanee
        scan_large(input, output, len);
    }
}

void transposer::reference::scan(int INPUT_ARRAY input, int * output, int len) {

    output[0] = 0;
    for(int i = 1; i < len; i++) {
        output[i] = output[i-1] + input[i-1];
    }
}

// ===============================================================================
// SEG SORT ======================================================================
// ===============================================================================

#define SEGSORT_ELEMENTS_PER_BLOCK 4
// #define SEGSORT_ELEMENTS_PER_BLOCK 32

__global__
void segsort_kernel(int INPUT_ARRAY input, int * output, int len) {

    __shared__ int temp[SEGSORT_ELEMENTS_PER_BLOCK];
    int thread_id = threadIdx.x;
    int global_id = blockIdx.x * SEGSORT_ELEMENTS_PER_BLOCK + threadIdx.x;

    // caricamento dati in shared memory
    int element = (global_id < len) ? input[global_id] : INT32_MAX;
    temp[thread_id] = element;
    __syncthreads();

    /// trovo la posizione del `thread_id`-esimo elemento
    int index = 0;
    for(int i = 0; i < thread_id; i++) {
        if(temp[i] <= element) {
            index++;
        }
    }
    for(int i = thread_id+1; i < SEGSORT_ELEMENTS_PER_BLOCK; i++) {
        if(temp[i] < element) {
            index++;
        }
    }
    __syncthreads();

    // porto alla posizione corretta
    temp[index] = element;
    __syncthreads();

    // scaricamento dati in shared memory
    if(thread_id < len) {
        output[global_id] = temp[thread_id];
    }
}

void transposer::cuda::segsort(int INPUT_ARRAY input, int * output, int len) {

    segsort_kernel<<< DIV_THEN_CEIL(len, SEGSORT_ELEMENTS_PER_BLOCK), SEGSORT_ELEMENTS_PER_BLOCK >>>(
        input, output, len
    );
    CUDA_CHECK_ERROR
}

void transposer::reference::segsort(int INPUT_ARRAY input, int * output, int len) {

    utils::copy_array(output, input, len);

    const int N = DIV_THEN_CEIL(len, SEGSORT_ELEMENTS_PER_BLOCK);
    for(int i = 0; i < N; i++) {
        const int start = i * SEGSORT_ELEMENTS_PER_BLOCK;
        const int end = std::min((i + 1) * SEGSORT_ELEMENTS_PER_BLOCK, len);
        std::sort(output + start, output + end); // TODO controlla comparatore default
    }
}

// ===============================================================================
// SEG SORT ======================================================================
// ===============================================================================

__global__
void merge_kernel(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {

    int global_id = blockIdx.x; // * SEGSORT_ELEMENTS_PER_BLOCK + threadIdx.x;
    int start_1 = 2 * global_id * BLOCK_SIZE;
    int start_2 = (2 * global_id + 1) * BLOCK_SIZE;
    int end_1 = min((2 * global_id + 1) * BLOCK_SIZE, len);
    int end_2 = min((2 * global_id + 2) * BLOCK_SIZE, len);

    //printf("%d: start1= %d, start2=%d, end1=%d, end2=%d\n", global_id, start_1, start_2, end_1, end_2);

    int current_1 = start_1;
    int current_2 = start_2;
    int current_output = start_1;
    
    // merge
    while(current_1 < end_1 && current_2 < end_2) {
        if(input[current_1] <= input[current_2]) {
            //printf("MERGE1 output[%d] = input[%d] = %d\n", current_output, current_1, input[current_1]);
            output[current_output] = input[current_1];
            current_1++;
        } else {
            //printf("MERGE2 output[%d] = input[%d] = %d\n", current_output, current_1, input[current_1]);
            output[current_output] = input[current_2];
            current_2++;
        }
        current_output++;
    }

    // finisco le rimanenze del primo blocco
    while(current_1 < end_1) {
        //printf("COPY1 output[%d] = input[%d] = %d\n", current_output, current_1, input[current_1]);
        output[current_output] = input[current_1];
        current_1++;
        current_output++;
    }

    // finisco le rimanenze del secondo blocco
    while(current_2 < end_2) {
        //printf("COPY2 output[%d] = input[%d] = %d\n", current_output, current_2, input[current_2]);
        output[current_output] = input[current_2];
        current_2++;
        current_output++;
    }
}

#define MERGE_SPLITTER_DISTANCE 4
// #define MERGE_SPLITTER_DISTANCE 256

__global__
void seg_splitter_kernel(int INPUT_ARRAY input, int * splitter, int len, const int BLOCK_SIZE) {

    int block_id = blockIdx.x;
    int th_id = threadIdx.x;

    // indici di inizio e fine della porzione di array di input
    int start_input = block_id * BLOCK_SIZE;
    // int end_input = min((block_id + 1) * BLOCK_SIZE, len);

    // indici di inizio e fine della porzione di array di splitter
    const int SPLITTER_BLOCK_SIZE = DIV_THEN_CEIL(BLOCK_SIZE, MERGE_SPLITTER_DISTANCE);
    int start_splitter = block_id * SPLITTER_BLOCK_SIZE;

    // riempio l'array di splitter
    for(int i = th_id; i < SPLITTER_BLOCK_SIZE; i = i + blockDim.x) {
        //int offset = i * MERGE_SPLITTER_DISTANCE;
        //if(start_input + offset <= end_input) {
            splitter[start_splitter + i] = input[start_input + i * MERGE_SPLITTER_DISTANCE];
        //} else {
        //    break;
        //}        
    }
}


__device__
int find(int element_to_search, int INPUT_ARRAY input, int len) {

    int start = 0;
    int end = start + len;

    while(start < end) {
        int current = (start + end) / 2;
        if(input[current] < element_to_search) {
            start = current + 1;
        } else if(input[current] >= element_to_search) {
            end = current;
        }
    }

    return start;
}

__global__
void seg_searchindex_kernel(int INPUT_ARRAY input, int INPUT_ARRAY splitter, int * indexA, int * indexB, int len, const int BLOCK_SIZE) {

    int block_id = blockIdx.x;
    int th_id = threadIdx.x;

    // indici di inizio e fine della porzione di array di input assegnata al blocco A, ed al blocco B
    int start_A = 2 * block_id * BLOCK_SIZE;
    int start_B = (2 * block_id + 1) * BLOCK_SIZE;
    int end_A = min((2 * block_id + 1) * BLOCK_SIZE, len);
    int end_B = min((2 * block_id + 2) * BLOCK_SIZE, len);

    // indici di inizio della porzione di array di splitter
    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, MERGE_SPLITTER_DISTANCE);
    int start_splitter = 2 * block_id       * SPLITTER_PER_BLOCK;
    int end_splitter   = 2 * (block_id + 1) * SPLITTER_PER_BLOCK;

    // ogni thread si dedica ad un elemento di splitter
    for(int i = start_splitter + th_id; i < end_splitter; i = i + blockDim.x) {
        int element = splitter[i];
        indexA[i] = find(element, input + start_A, end_A - start_A);
        indexB[i] = find(element, input + start_B, end_B - start_B);
    }
}

__global__
void seg_splittermerge_kernel(int INPUT_ARRAY input, int INPUT_ARRAY indexA, int INPUT_ARRAY indexB, int * output, int len, const int BLOCK_SIZE) {

    // seg_splittermerge_kernel<<<BLOCK_NUMBER * (SPLITTER_PER_BLOCK + 1), MERGE_SPLITTER_DISTANCE>>>

    int i = blockIdx.x;

    // Ogni `i` (blockIdx.x) effettua un piccolo merge in memoria shared. 
    // Devo risalire a quale SUB-SPLITTER corrisponde l'elemento `i` per sapere se è
    // il primo o l'ultimo del SUB-SPLITTER 
    
    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, MERGE_SPLITTER_DISTANCE);
    const int SPLITTER_PER_COUPLE_BLOCKS = 2 * SPLITTER_PER_BLOCK + 1;

    int which_item = i % SPLITTER_PER_COUPLE_BLOCKS;
    bool is_first_element = (which_item == 0);
    bool is_last_element = (which_item == SPLITTER_PER_COUPLE_BLOCKS-1);

    // prendo gli indici di inizio e fine del blocco in input contrassegnato 
    // da A, e B. Se sono il primo blocco parto da 0, se sono l'ultimo finisco con BLOCK_SIZE
    int startA = is_first_element ? 0 : indexA[i-1];
    int startB = is_first_element ? 0 : indexB[i-1];
    int endA   = is_last_element ? BLOCK_SIZE : indexA[i];
    int endB   = is_last_element ? BLOCK_SIZE : indexB[i];

    // Ogni griglia di thread ha una memoria condivisa sulla quale copiare gli elementi
    // dell'array di input fino ad un massimo di MERGE_SPLITTER_DISTANCE per blocco
    __shared__ int temp[2 * MERGE_SPLITTER_DISTANCE];

    // Carico le porzioni degli elementi
    int j = threadIdx.x;
    // elemento in input preso dall'elemento A della coppia di blocchi
    temp[j] = (startA + j < endA) ? input[startA + j] : 0;
    // elemento in input preso dall'elemento B della coppia di blocchi
    temp[MERGE_SPLITTER_DISTANCE + j] = (startB + j < endB) ? input[startB + j] : 0;
    // aspetto che temp sia stato completamente caricato
    __syncthreads();

    // cerco l'indice del mio elemento preso da A nella coppia di blocchi A, B mergiati
    int index1 = j + find(temp[j], temp + MERGE_SPLITTER_DISTANCE, endB - startB);       
    // cerco l'indice del mio elemento preso da B nella coppia di blocchi A, B mergiati
    int index2 = find(temp[MERGE_SPLITTER_DISTANCE + j], temp, endA - startA) + (j + MERGE_SPLITTER_DISTANCE); // elemento di B
    // aspetto che tutti abbiano cercato in temp
    __syncthreads();

    // salvo in output (start_A è l'inizio dell'intera coppia di blocchi)
    output[startA + index1] = temp[j];
    output[startA + index2] = temp[MERGE_SPLITTER_DISTANCE + j];

    printf("Block=%d, Thread=%d, A=[%d:%d], B=[%d:%d], elementA=%d->%d, elementB=%d->%d\n", 
        blockIdx.x, j, startA, endA, startB, endB, temp[j], index1, temp[MERGE_SPLITTER_DISTANCE + j], index2
    );
}

void merge(int INPUT_ARRAY input, int * output, int len, const int BLOCK_SIZE);

void merge_big(int INPUT_ARRAY input, int * output, int len, const int BLOCK_SIZE) {

    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, MERGE_SPLITTER_DISTANCE);
    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    const int SPLITTER_NUMBER = BLOCK_NUMBER * SPLITTER_PER_BLOCK;

    int * splitter        = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * splitter_sorted = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA          = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB          = utils::cuda::allocate<int>(SPLITTER_NUMBER);

    // 1. faccio lo split degli elementi 
    // Uso una grid per ogni sub-splitter array, ogni griglia può avere il numero di thread che mi pare per dividere il lavoro di copia
    seg_splitter_kernel<<<BLOCK_SIZE, 32>>>(input, splitter, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR
    DPRINT_ARR_CUDA(splitter, DIV_THEN_CEIL(len, MERGE_SPLITTER_DISTANCE))

    // 2. faccio il merge degli elementi di splitter
    // Faccio il merge di `splitter` che è lungo SPLITTER_NUMBER elementi
    // divisi in BLOCK_NUMBER blocchi da SPLITTER_PER_BLOCK elementi.
    DPRINT_MSG("Merging splitter with len=%d, block_size=%d", SPLITTER_NUMBER, SPLITTER_PER_BLOCK)
    merge(splitter, splitter_sorted, SPLITTER_NUMBER, SPLITTER_PER_BLOCK);
    DPRINT_ARR_CUDA(splitter_sorted, SPLITTER_PER_BLOCK)

    // 3. cerco gli indici di ogni elemento
    // Utilizzo una GRIGLIA[di thread] ogni 2 blocchi.
    // Ogni griglia ha un numero arbitrario (es. 32) di thread che ci lavorano
    seg_searchindex_kernel<<<DIV_THEN_CEIL(BLOCK_SIZE, 2), 32>>>(input, splitter_sorted, indexA, indexB, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR
    DPRINT_ARR_CUDA(indexA, DIV_THEN_CEIL(len, MERGE_SPLITTER_DISTANCE))
    DPRINT_ARR_CUDA(indexB, DIV_THEN_CEIL(len, MERGE_SPLITTER_DISTANCE))

    // 4. ogni thread fa ora il merge delle porzioni di A, B associate a splitter[i]
    // Ogni GRIGLIA[di thread] processa il merge di un numero massimo di elementi. 
    // Devo avere BLOCK_NUMBER * (SPLITTER_PER_BLOCK + 1) griglie perchè ad ogni blocco
    // sono associato SPLITTER_PER_BLOCK segmenti + uno finale
    // La griglia di thread viene dimensionata MERGE_SPLITTER_DISTANCE così che ogni thread
    // processa esattamente un elemento
    const int SPLITTER_PER_COUPLE_BLOCKS = 2 * SPLITTER_PER_BLOCK + 1;
    const int COUPLE_OF_BLOCKS = DIV_THEN_CEIL(BLOCK_NUMBER, 2);

    seg_splittermerge_kernel<<<COUPLE_OF_BLOCKS * (SPLITTER_PER_COUPLE_BLOCKS + 1), MERGE_SPLITTER_DISTANCE>>>(
        input, indexA, indexB, output, len, BLOCK_SIZE
    );
    CUDA_CHECK_ERROR
    DPRINT_ARR_CUDA(output, len)

    utils::cuda::deallocate(splitter);
    utils::cuda::deallocate(indexA);
    utils::cuda::deallocate(indexB);
}

void merge(int INPUT_ARRAY input, int * output, int len, const int INIT_BLOCK_SIZE) {

    int* buffer[2];
    buffer[0] = output;
    buffer[1] = utils::cuda::allocate<int>(len);
    int full = 0;

    // applico merge
    for(int BLOCK_SIZE = INIT_BLOCK_SIZE; BLOCK_SIZE < len; BLOCK_SIZE *= 2) {
        
        DPRINT_MSG("Block size=%d", BLOCK_SIZE)
        if(BLOCK_SIZE > 1) {
merge_big(buffer[full], buffer[1-full], len, BLOCK_SIZE);
        }
        

        full = 1 - full;
        DPRINT_ARR_CUDA(buffer[full], len)
    }

    // eventualmente copio nell'array di output nel caso non sia stato l'ultimo
    // ad essere riempito...
    if(full != 0) {
        utils::cuda::copy(output, buffer[1], len);
    }

    // dealloco array temporaneo;
    utils::cuda::deallocate(buffer[1]);



    //if(2 * BLOCK_SIZE > 2 * MERGE_SPLITTER_DISTANCE) {
    //    // due blocchi non rientrano nella shared memory allora uso il metodo `big merge`
    //    DPRINT_MSG("Block size=%d", BLOCK_SIZE)
    //    merge_big(input, output, len, BLOCK_SIZE);
    //
    //} else {
    //    // entrambi i blocchi insieme stanno nella shared memory
    //    DPRINT_MSG("Small merge: block size=%d", BLOCK_SIZE)
    //    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    //    merge_kernel<<< DIV_THEN_CEIL(BLOCK_NUMBER, 2), 1 >>>(input, output, len, BLOCK_SIZE);
    //    CUDA_CHECK_ERROR
    //    DPRINT_ARR_CUDA(output, len)
    //}
}

void transposer::cuda::sort(int INPUT_ARRAY input, int * output, int len) {

    transposer::cuda::segsort(input, output, len);

    merge(input, output, len, SEGSORT_ELEMENTS_PER_BLOCK);
}

void transposer::reference::sort(int INPUT_ARRAY input, int * output, int len) {

    utils::copy_array(output, input, len);
    std::sort(output, output + len);
}

// ===============================================================================
// SERIAL IMPLEMENTATION =========================================================
// ===============================================================================

int transposer::reference::serial_csr2csc(
    int m, int n, int nnz, 
    int INPUT_ARRAY csrRowPtr, int INPUT_ARRAY csrColIdx, float INPUT_ARRAY csrVal, 
    int *cscColPtr, int *cscRowIdx, float *cscVal
) {
    // 1. costruisco `cscColPtr` come istogramma delle frequenze degli elementi per ogni colonna
    DPRINT_MSG("Step 1: idx to ptr")
    indexes_to_pointers(csrColIdx, nnz, cscColPtr, n+1);

    // 2. applico prefix_sum per costruire corretto `cscColPtr` (ogni cella tiene conto dei precedenti)
    DPRINT_MSG("Step 2: prefix sum")
    utils::prefix_sum(cscColPtr, n+1);

    // 3. sistemo indici di riga e valori
    DPRINT_MSG("Step 3: fix row, value arrays")
    int* curr = new int[n](); 

    for(int i = 0; i < m; i++) {
        for(int j = csrRowPtr[i]; j < csrRowPtr[i+1]; j++) {
            int col = csrColIdx[j];
            int loc = cscColPtr[col] + curr[col];
            curr[col]++;
            cscRowIdx[loc] = i;
            cscVal[loc] = csrVal[j];
        }
    }

    DPRINT_MSG("End")

    delete[] curr;
    return COMPUTATION_OK;
}


// ===============================================================================
// COMPONENT TEST ================================================================
// ===============================================================================

bool transposer::component_test::indexes_to_pointers() {

    const int N = 10000, NNZ = 10000;
    // input
    int *idx = utils::random::generate_array(0, N-1, NNZ);
    DPRINT_ARR(idx, NNZ)

    // reference implementation
    int *ptr = new int[N+1];
    transposer::reference::indexes_to_pointers(idx, NNZ, ptr, N+1);
    DPRINT_ARR(ptr, N+1)

    // cuda implementation
    int *idx_cuda = utils::cuda::allocate_send<int>(idx, NNZ);
    int *ptr_cuda = utils::cuda::allocate_zero<int>(N+1);
    transposer::cuda::indexes_to_pointers(idx_cuda, NNZ, ptr_cuda, N+1);
    int *ptr2 = new int[N+1]; utils::cuda::recv(ptr2, ptr_cuda, N+1);
    DPRINT_ARR(ptr2, N+1)

    bool ok = utils::equals<int>(ptr, ptr2, N+1);

    utils::cuda::deallocate(idx_cuda);
    utils::cuda::deallocate(ptr_cuda);
    delete idx, ptr, ptr2;
    
    return ok;
} 

bool transposer::component_test::pointers_to_indexes() {

    const int N = 10000, NNZ = 10000;

    int *ptr = utils::random::generate_array(0, 1, N+1);
    ptr[N] = 0;
    utils::prefix_sum(ptr, N+1);
    DPRINT_ARR(ptr, N+1)

    // reference implementation
    int *idx = new int[NNZ];
    reference::pointers_to_indexes(ptr, N+1, idx, NNZ);
    DPRINT_ARR(idx, NNZ)

    // cuda implementation
    int *ptr_cuda = utils::cuda::allocate_send<int>(ptr, N+1);
    int *idx_cuda = utils::cuda::allocate_zero<int>(NNZ);
    transposer::cuda::pointers_to_indexes(ptr_cuda, N+1, idx_cuda, NNZ);
    int *idx2 = new int[N+1]; utils::cuda::recv(idx2, idx_cuda, NNZ);
    DPRINT_ARR(idx2, NNZ)

    bool ok = utils::equals<int>(idx, idx2, NNZ);

    utils::cuda::deallocate(idx_cuda);
    utils::cuda::deallocate(ptr_cuda);
    delete ptr, idx, idx2;
    
    return ok;
}

bool transposer::component_test::scan() {

    const int N = 1000000;
    // input
    int *arr = utils::random::generate_array(1, 1, N);
    DPRINT_ARR(arr, N)

    // reference implementation
    int *scan_arr = new int[N];
    transposer::reference::scan(arr, scan_arr, N);
    DPRINT_ARR(scan_arr, N)

    // cuda implementation
    int *scan_arr_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
    int *scan_arr_cuda_out = utils::cuda::allocate_zero<int>(N);
    transposer::cuda::scan(scan_arr_cuda_in, scan_arr_cuda_out, N);
    int *scan_arr_2 = new int[N]; 
    utils::cuda::recv(scan_arr_2, scan_arr_cuda_out, N);
    DPRINT_ARR(scan_arr_2, N)

    bool ok = utils::equals<int>(scan_arr, scan_arr_2, N);

    utils::cuda::deallocate(scan_arr_cuda_in);
    utils::cuda::deallocate(scan_arr_cuda_out);
    delete arr, scan_arr, scan_arr_2;
    
    return ok;
} 

bool transposer::component_test::segsort() {

    const int N = 10000000;
    // input
    int *arr = utils::random::generate_array(1, 100, N);
    DPRINT_ARR(arr, N)

    // reference implementation
    int *segsort_arr = new int[N];
    transposer::reference::segsort(arr, segsort_arr, N);
    DPRINT_ARR(segsort_arr, N)

    // cuda implementation
    int *segsort_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
    int *segsort_cuda_out = utils::cuda::allocate<int>(N);
    transposer::cuda::segsort(segsort_cuda_in, segsort_cuda_out, N);
    int *segsort_arr_2 = new int[N]; 
    utils::cuda::recv(segsort_arr_2, segsort_cuda_out, N);
    DPRINT_ARR(segsort_arr_2, N)

    bool ok = utils::equals<int>(segsort_arr, segsort_arr_2, N);

    utils::cuda::deallocate(segsort_cuda_in);
    utils::cuda::deallocate(segsort_cuda_out);
    delete arr, segsort_arr, segsort_arr_2;
    
    return ok;
}

bool transposer::component_test::sort() {

    const int N = 10;
    // input
    int *arr = utils::random::generate_array(1, 100, N);
    DPRINT_ARR(arr, N)

    // reference implementation
    int *sort_arr = new int[N];
    transposer::reference::sort(arr, sort_arr, N);

    // cuda implementation
    int *sort_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
    int *sort_cuda_out = utils::cuda::allocate<int>(N);
    transposer::cuda::sort(sort_cuda_in, sort_cuda_out, N);
    int *sort_arr_2 = new int[N]; 
    utils::cuda::recv(sort_arr_2, sort_cuda_out, N);

    DPRINT_ARR(sort_arr, N)
    DPRINT_ARR(sort_arr_2, N)
    bool ok = utils::equals<int>(sort_arr, sort_arr_2, N);

    utils::cuda::deallocate(sort_cuda_in);
    utils::cuda::deallocate(sort_cuda_out);
    delete arr, sort_arr, sort_arr_2;
    
    return ok;
}