#include "hip/hip_runtime.h"
#include "procedures.hh"

#define BLOCK_DIM 16

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void transpose(int *odata, int *idata, int width, int height)
{
	__shared__ int block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
    // load one element per thread from device memory (idata) and store it
    // in transposed order in block[][]
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

    // synchronise to ensure all writes to block[][] have completed
	__syncthreads();

	// write the transposed matrix tile to global memory (odata) in linear order
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

void procedures::cuda::transpose(int INPUT_ARRAY input, int * output, int width, int height) {

    if(width % BLOCK_DIM || height % BLOCK_DIM) {
        std::cerr << "Size is not correct\n";
        return;
    }

    dim3 grid(size_x / BLOCK_DIM, size_y / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    transpose<<< grid, threads >>>(input, output, width, height);
    CUDA_CHECK_ERROR
}

void procedures::reference::transpose(int INPUT_ARRAY input, int * output, int width, int height) {
    for(int i = 0; i < width; i++) {
        for(int j = 0; j < height; j++) {
            input[i * width + j] = output[j * width + i];
        }
    }
}
