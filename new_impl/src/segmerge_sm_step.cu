#include "hip/hip_runtime.h"
#include "transposer.hh"

__device__
void copy(int * output, int INPUT_ARRAY input, int len) {
    for(int i = 0; i < len; i++) {
        output[i] = input[i];
    }
}

__device__
int binary_search(int element_to_search, int INPUT_ARRAY input, int len) {

    int start = 0;
    int end = start + len;

    while(start < end) {
        int current = (start + end) / 2;
        if(input[current] < element_to_search) {
            start = current + 1;
        } else if(input[current] >= element_to_search) {
            end = current;
        }
    }

    return start;
}

__device__
int binary_search_last(int element_to_search, int INPUT_ARRAY input, int len) {

    int index = binary_search(element_to_search, input, len);
    while(input[index] == element_to_search && index < len) {
        index++;
    }
    return index;
}

__global__
void splitter_kernel(int INPUT_ARRAY input, int * splitter, int * indexA, int * indexB, int len, int BLOCK_SIZE) {

    int couple_block_id = blockIdx.x;

    // entrambi inputA, inputB esistono (eventualmente B ha lunghezza < BLOCK_SIZE se ultimo blocco)
    int * inputA = input + 2 * couple_block_id * BLOCK_SIZE;
    int * inputB = input + (2 * couple_block_id + 1) * BLOCK_SIZE;
    int lenA = BLOCK_SIZE;
    int endB = min((2 * couple_block_id + 2) * BLOCK_SIZE, len);
    int lenB = endB - (2 * couple_block_id + 1) * BLOCK_SIZE;

    // mi sposto verso gli indici corretti di splitter, indexA, indexB
    const int SPLITTER_PER_BLOCKS = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    splitter = splitter + 2 * couple_block_id * SPLITTER_PER_BLOCKS;
    indexA = indexA + 2 * couple_block_id * SPLITTER_PER_BLOCKS;
    indexB = indexB + 2 * couple_block_id * SPLITTER_PER_BLOCKS;

    // riempio gli elementi
    int i;
    for(int i = 0; i < SPLITTER_PER_BLOCKS && i*SEGMERGE_SM_SPLITTER_DISTANCE < lenA; i++) {
        splitter[i] = inputA[i*SEGMERGE_SM_SPLITTER_DISTANCE];
        if(i > 0) {
            // shifto indietro di 1 per evitare il primo merge (di porzioni di array vuote)
            indexA[i - 1] = i*SEGMERGE_SM_SPLITTER_DISTANCE;
            indexB[i - 1] = binary_search(splitter[i], inputB, lenB);
        }
    }

    for(i = 0; i < SPLITTER_PER_BLOCKS && i*SEGMERGE_SM_SPLITTER_DISTANCE < lenB; i++) {
        splitter[SPLITTER_PER_BLOCKS + i] = inputB[i*SEGMERGE_SM_SPLITTER_DISTANCE];
        // shifto indietro di 1 per evitare il primo merge (di porzioni di array vuote)
        indexA[SPLITTER_PER_BLOCKS + i - 1] = binary_search_last(splitter[i], inputB, lenB);
        indexB[SPLITTER_PER_BLOCKS + i - 1] = i*SEGMERGE_SM_SPLITTER_DISTANCE;
    }

    // alla fine degli indici ci sono le dimensioni degli array
    indexA[SPLITTER_PER_BLOCKS + i - 1] = lenA;
    indexB[SPLITTER_PER_BLOCKS + i - 1] = lenB;
}

__global__
void uniform_merge_kernel(int INPUT_ARRAY input, int * output, int INPUT_ARRAY indexA, int INPUT_ARRAY indexB, int len, int BLOCK_SIZE) {

    __shared__ int temp_in[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ int temp_out[2 * SEGMERGE_SM_SPLITTER_DISTANCE];

    // processa l'elemento dello splitter
    const int splitter_index = blockIdx.x;

    // recupera blocco sul quale stai lavorando
    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    int couple_block_id = splitter_index / (2 * SPLITTER_PER_BLOCK);
    int item = splitter_index % (2 * SPLITTER_PER_BLOCK);

    // recupera estremi sui quali lavorare
    int * inputA = input + 2 * couple_block_id * BLOCK_SIZE;
    int * inputB = input + (2 * couple_block_id + 1) * BLOCK_SIZE;
    int startA = (item == 0) ? 0 : indexA[splitter_index-1];
    int startB = (item == 0) ? 0 : indexB[splitter_index-1];
    int endA   = indexA[splitter_index];
    int endB   = indexB[splitter_index];

    // carico gli elementi in temp_in
    copy(temp_in, inputA + startA, endA - startA);
    copy(temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, inputB + startB, endB - startB);

    // effettuo merge
    for(int i = 0; i < endA - startA; i++) {
        int k = i + binary_search(temp_in[i], temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, endB - startB);
        temp_out[k] = temp_in[i];
    }
    for(int i = 0; i < endB - startB; i++) {
        int element = temp_in[SEGMERGE_SM_SPLITTER_DISTANCE + i];
        int k = i + binary_search_last(element, temp_in, endA - startA);
        temp_out[k] = temp_in[i];
    }

    // salva output
    output = output + 2 * couple_block_id * BLOCK_SIZE;
    for(int i = 0; i < (endA - startA) + (endB - startB); i++) {
        output[startA + startB + i] = temp_out[i];
    }
}

void transposer::cuda::segmerge_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {
    
    // 1. lavoro su coppie di blocchi per estrarre gli splitter e gli indici necessari a lavorarci sopra
    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    const int COUPLE_OF_BLOCKS = BLOCK_NUMBER / 2;
    const int SPLITTER_PER_BLOCKS = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    const int SPLITTER_NUMBER = 2 * COUPLE_OF_BLOCKS * SPLITTER_PER_BLOCKS;

    int * splitter     = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * splitter_out = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);

    splitter_kernel<<<COUPLE_OF_BLOCKS, 1>>>(input, splitter, indexA, indexB, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR
    DPRINT_MSG("After splitter_kernel")
    DPRINT_ARR_CUDA(splitter, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexA, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexB, SPLITTER_NUMBER)

    // 2. riordino per blocchi l'array degli splitter e gli indici ad esso associati
    segmerge3_sm_step(splitter, splitter_out, SPLITTER_NUMBER, SPLITTER_PER_BLOCKS, indexA, indexA_out, indexB, indexB_out);
    DPRINT_MSG("After segmerge3_sm_step")
    DPRINT_ARR_CUDA(splitter_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexA_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexB_out, SPLITTER_NUMBER)

    // 3. eseguo il merge di porzioni di blocchi di dimensione uniforme
    uniform_merge_kernel<<<SPLITTER_NUMBER, 1>>>(input, output, indexA_out, indexB_out, len, BLOCK_SIZE);
    DPRINT_MSG("After uniform_merge_kernel")
    DPRINT_ARR_CUDA(input, len)
    DPRINT_ARR_CUDA(output, len)
    CUDA_CHECK_ERROR

    // 4. eventualmente copio il risultato dell' ultimo blocco di array rimasto spaiato
    if(BLOCK_NUMBER % 2 == 1) {
        utils::cuda::copy<int>(
            output + 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            input +  2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            len - 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE
        );
    }

    utils::cuda::deallocate(splitter);
    utils::cuda::deallocate(indexA);
    utils::cuda::deallocate(indexB);
    utils::cuda::deallocate(splitter_out);
    utils::cuda::deallocate(indexA_out);
    utils::cuda::deallocate(indexB_out);
}

void transposer::cuda::segmerge3_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, int INPUT_ARRAY b_in, int * b_out) {
    segmerge3_step(input, output, len, BLOCK_SIZE, a_in, a_out, b_in, b_out);
}

void transposer::reference::segmerge_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {
    segmerge_step(input, output, len, BLOCK_SIZE);
}

void transposer::reference::segmerge3_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, int INPUT_ARRAY b_in, int * b_out) {
    segmerge3_step(input, output, len, BLOCK_SIZE, a_in, a_out, b_in, b_out);
}


// ================================================
//  segmerge sm 
bool transposer::component_test::segmerge_sm_step() {

    const int N = 100;
    int BLOCK_SIZE = 4;
    // input

    bool oks = true;
    for(int j=0; j < 10; j++){

        int rand_value = utils::random::generate(0,12345);
        int *arr = utils::random::generate_array(1 + rand_value ,100 + rand_value, N);
        if(j<8) BLOCK_SIZE *= 2;
        DPRINT_ARR(arr, N)

        // reference implementation
        int *segmerge_sm_arr = new int[N];
        transposer::reference::segmerge_sm_step(arr, segmerge_sm_arr, N, BLOCK_SIZE);
        DPRINT_ARR(segmerge_sm_arr, N)

        // cuda implementation
        int *segmerge_sm_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
        int *segmerge_sm_cuda_out = utils::cuda::allocate<int>(N);
        transposer::cuda::segmerge_sm_step(segmerge_sm_cuda_in, segmerge_sm_cuda_out, N, BLOCK_SIZE);
        int *segmerge_sm_arr_2 = new int[N]; 
        utils::cuda::recv(segmerge_sm_arr_2, segmerge_sm_cuda_out, N);
        DPRINT_ARR(segmerge_sm_arr_2, N)

        bool ok = utils::equals<int>(segmerge_sm_arr, segmerge_sm_arr_2, N);
        oks = oks && ok;
        utils::cuda::deallocate(segmerge_sm_cuda_in);
        utils::cuda::deallocate(segmerge_sm_cuda_out);
        delete arr, segmerge_sm_arr, segmerge_sm_arr_2;


    }
    return oks;
}

// ================================================
//  segmerge3 sm 
bool transposer::component_test::segmerge3_sm_step() {

    const int N = 100;
    // input

    bool oks = true;
    int BLOCK_SIZE = 4;

    for(int j=0; j < 10; j++){
        int rand_value = utils::random::generate(0,12345);
        int *arr = utils::random::generate_array(1 + rand_value ,100 + rand_value, N);
        if(j<8) BLOCK_SIZE *= 2;
        DPRINT_ARR(arr, N)

        // reference implementation
        int *segmerge_sm_arr = new int[N];
        int *segmerge_a_in_arr = new int[N];
        int *segmerge_a_out_arr = new int[N];
        int *segmerge_b_in_arr = new int[N];
        int *segmerge_b_out_arr = new int[N];
        transposer::reference::segmerge3_sm_step(arr, segmerge_sm_arr, N, BLOCK_SIZE,
                                                segmerge_a_in_arr, segmerge_a_out_arr, segmerge_b_in_arr, segmerge_b_out_arr);

        DPRINT_ARR(segmerge_sm_arr, N)

        // cuda implementation
        int *segmerge_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
        int *segmerge_a_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
        int *segmerge_b_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
        int *segmerge_cuda_out = utils::cuda::allocate<int>(N);
        int *segmerge_a_cuda_out = utils::cuda::allocate<int>(N);
        int *segmerge_b_cuda_out = utils::cuda::allocate<int>(N);
    
        transposer::cuda::segmerge3_sm_step(segmerge_cuda_in, segmerge_cuda_out, N, BLOCK_SIZE,
                                            segmerge_a_cuda_in, segmerge_b_cuda_in, segmerge_a_cuda_out, segmerge_b_cuda_out);
        int *segmerge_sm_arr_2 = new int[N]; 
        utils::cuda::recv(segmerge_sm_arr_2, segmerge_cuda_out, N);
        DPRINT_ARR(segmerge_sm_arr_2, N)

        bool ok = utils::equals<int>(segmerge_sm_arr, segmerge_sm_arr_2, N);
        oks = oks && ok;
        utils::cuda::deallocate(segmerge_cuda_in);
        utils::cuda::deallocate(segmerge_a_cuda_in);
        utils::cuda::deallocate(segmerge_b_cuda_in);
        utils::cuda::deallocate(segmerge_cuda_out);
        utils::cuda::deallocate(segmerge_a_cuda_out);
        utils::cuda::deallocate(segmerge_b_cuda_out);
        delete arr, segmerge_sm_arr, segmerge_sm_arr_2, segmerge_a_in_arr, segmerge_a_out_arr, segmerge_b_in_arr, segmerge_b_out_arr;

    }
    return oks;
}

/*
// ===============================================================================
bool transposer::component_test::segmerge_static_sm() {

    const int N = 49;

    int *arr = utils::random::generate_array(1, 5, N);
    DPRINT_ARR(arr, N)

    // reference implementation
    int *segmerge_arr = new int[N];
    transposer::reference::segmerge_step(arr, segmerge_arr, N);
    DPRINT_ARR(segmerge_arr, N)

    // cuda implementation
    int *segmerge_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
    int *segmerge_cuda_out = utils::cuda::allocate<int>(N);
    transposer::cuda::segmerge3_step(segmerge_cuda_in, segmerge_cuda_out, N);
    int *segmerge_arr_2 = new int[N]; 
    utils::cuda::recv(segmerge_arr_2, segmerge_cuda_out, N);
    DPRINT_ARR(segmerge_arr_2, N)

    bool ok = utils::equals<int>(segmerge_arr, segmerge_arr_2, N);

    utils::cuda::deallocate(segmerge_cuda_in);
    utils::cuda::deallocate(segmerge_cuda_out);
    delete arr, segmerge_arr, segmerge_arr_2;

    return ok;
}


// ===============================================================================
bool transposer::component_test::segmerge3_static_sm() {

    const int N = 49;

    int *arr = utils::random::generate_array(1, 5, N);
    DPRINT_ARR(arr, N)

    // reference implementation
    int *segmerge_arr = new int[N];
    transposer::reference::segmerge_step(arr, segmerge_arr, N);
    DPRINT_ARR(segmerge_arr, N)

    // cuda implementation
    int *segmerge_cuda_in  = utils::cuda::allocate_send<int>(arr, N);
    int *segmerge_cuda_out = utils::cuda::allocate<int>(N);
    transposer::cuda::segmerge3_step(segmerge_cuda_in, segmerge_cuda_out, N);
    int *segmerge_arr_2 = new int[N]; 
    utils::cuda::recv(segmerge_arr_2, segmerge_cuda_out, N);
    DPRINT_ARR(segmerge_arr_2, N)

    bool ok = utils::equals<int>(segmerge_arr, segmerge_arr_2, N);

    utils::cuda::deallocate(segmerge_cuda_in);
    utils::cuda::deallocate(segmerge_cuda_out);
    delete arr, segmerge_arr, segmerge_arr_2;

    return ok;
}
*/