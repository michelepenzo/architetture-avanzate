#include "hip/hip_runtime.h"
#include "procedures.hh"

__device__
int find_position_in_sorted_array(int element_to_search, int INPUT_ARRAY input, int len) {

    if(len <= 0) {
        return 0;
    }

    int start = 0;
    int end = start + len;

    while(start < end) {
        int current = (start + end) / 2;
        if(input[current] < element_to_search) {
            start = current + 1;
        } else if(input[current] >= element_to_search) {
            end = current;
        }
    }

    return start;
}

__device__
int find_last_position_in_sorted_array(int element_to_search, int INPUT_ARRAY input, int len) {

    int index = find_position_in_sorted_array(element_to_search, input, len);
    while(input[index] == element_to_search && index < len) {
        //printf("input[%d] = %d == %d, index++\n", index, input[index], element_to_search);
        index++;
    }
    return index;
}

__global__
void splitter_kernel(int INPUT_ARRAY input, int * splitter, int * indexA, int * indexB, int len, int BLOCK_SIZE) {

    const int couple_block_id = blockIdx.x;
    const int thid = threadIdx.x;

    // entrambi inputA, inputB esistono (eventualmente B ha lunghezza < BLOCK_SIZE se ultimo blocco)
    int * inputA = input + 2 * couple_block_id * BLOCK_SIZE;
    int * inputB = input + (2 * couple_block_id + 1) * BLOCK_SIZE;
    int lenA = BLOCK_SIZE;
    int lenB = min((2 * couple_block_id + 2) * BLOCK_SIZE, len) - (2 * couple_block_id + 1) * BLOCK_SIZE;

    // mi sposto verso gli indici corretti di splitter, indexA, indexB
    const int SPLITTER_PER_BLOCKS = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    splitter = splitter + 2 * couple_block_id * SPLITTER_PER_BLOCKS;
    indexA = indexA + 2 * couple_block_id * SPLITTER_PER_BLOCKS;
    indexB = indexB + 2 * couple_block_id * SPLITTER_PER_BLOCKS;

    // riempio gli elementi
    int i;
    for(int i = thid; i < SPLITTER_PER_BLOCKS; i += blockDim.x) {
        splitter[i] = inputA[i*SEGMERGE_SM_SPLITTER_DISTANCE];
        indexA[i] = i*SEGMERGE_SM_SPLITTER_DISTANCE;
        indexB[i] = find_position_in_sorted_array(splitter[i], inputB, lenB);
    }
    __syncthreads();

    for(i = thid; i < SPLITTER_PER_BLOCKS && i*SEGMERGE_SM_SPLITTER_DISTANCE < lenB; i += blockDim.x) {
        int element = inputB[i*SEGMERGE_SM_SPLITTER_DISTANCE];
        // save splitter
        splitter[SPLITTER_PER_BLOCKS + i] = element;
        indexA[SPLITTER_PER_BLOCKS + i] = find_last_position_in_sorted_array(element, inputA, lenA);
        indexB[SPLITTER_PER_BLOCKS + i] = i*SEGMERGE_SM_SPLITTER_DISTANCE;
        //printf("(%2d): element %d from B is position %d of A\n", couple_block_id, element, indexA[SPLITTER_PER_BLOCKS + i]);
    }
}

__global__
void fix_indexes_kernel(int * indexA, int * indexB, int len, int BLOCK_SIZE, int SPLITTER_NUMBER, int SPLITTER_PER_BLOCK) {

    int couple_block_id = blockIdx.x;

    if(threadIdx.x == 0) {

        // calcolo gli indici di inizio e fine degli splitter che devo processare
        int startSplitter = 2 * couple_block_id * SPLITTER_PER_BLOCK;
        int endSplitter  = min(2 * (couple_block_id + 1) * SPLITTER_PER_BLOCK, SPLITTER_NUMBER);
        int lenSplitter = endSplitter - startSplitter;
        // la lunghezza di A è sempre BLOCK_SIZE, la lunghezza di B?... dipende se è l'ultimo
        int lenB = min(2 * (couple_block_id + 1) * BLOCK_SIZE, len) - ((2 * couple_block_id + 1) * BLOCK_SIZE);
        
        // printf("(%2d) split[%3d:%3d] {max %d}\n", blockIdx.x, startSplitter, endSplitter, SPLITTER_NUMBER);
    
        for(int i = 0; i < lenSplitter - 1; i += 1) {
            indexA[startSplitter+i] = indexA[startSplitter+i+1];
            indexB[startSplitter+i] = indexB[startSplitter+i+1];
        }

        // l'ultimo elemento contiene la dimensione del blocco
        indexA[startSplitter+lenSplitter-1] = BLOCK_SIZE;
        indexB[startSplitter+lenSplitter-1] = lenB;
        __syncthreads();
    }
}

__global__
void uniform_merge_kernel(int INPUT_ARRAY input, int * output, int INPUT_ARRAY indexA, int INPUT_ARRAY indexB, int len, int BLOCK_SIZE) {

    __shared__ int temp_in[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ int temp_out[2 * SEGMERGE_SM_SPLITTER_DISTANCE];

    for(int i = 0; i < 2 * SEGMERGE_SM_SPLITTER_DISTANCE; i++) {
        temp_in[i] = 0;
        temp_out[i] = 0;
    }

    // processa l'elemento dello splitter
    const int splitter_index = blockIdx.x;

    // recupera blocco sul quale stai lavorando
    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    int couple_block_id = splitter_index / (2 * SPLITTER_PER_BLOCK);
    int item = splitter_index % (2 * SPLITTER_PER_BLOCK);

    // recupera estremi sui quali lavorare
    int * inputA = input + 2 * couple_block_id * BLOCK_SIZE;
    int * inputB = input + (2 * couple_block_id + 1) * BLOCK_SIZE;
    int startA = (item == 0) ? 0 : indexA[splitter_index-1];
    int startB = (item == 0) ? 0 : indexB[splitter_index-1];
    int endA   = indexA[splitter_index];
    int endB   = indexB[splitter_index];
    
    // carico gli elementi in temp_in
    if(endA - startA > SEGMERGE_SM_SPLITTER_DISTANCE) printf("!!!Error A[%d:%d] > %d SM SP S\n", startA, endA, SEGMERGE_SM_SPLITTER_DISTANCE);
    utils::cuda::devcopy<int>(temp_in, inputA + startA, endA - startA);
    if(endB - startB > SEGMERGE_SM_SPLITTER_DISTANCE) printf("!!!Error B[%d:%d] > %d SM SP S\n", startB, endB, SEGMERGE_SM_SPLITTER_DISTANCE);
    utils::cuda::devcopy<int>(temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, inputB + startB, endB - startB);
    __syncthreads();

    // effettuo merge
    for(int i = 0; i < endA - startA; i++) {
        int element = temp_in[i];
        int posInA = i;
        int posInB = find_position_in_sorted_array(element, temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, endB - startB);
        int k = posInA + posInB;
        temp_out[k] = element;
    }
    __syncthreads();

    for(int i = 0; i < endB - startB; i++) {
        int element = temp_in[SEGMERGE_SM_SPLITTER_DISTANCE + i];
        int k = i + find_last_position_in_sorted_array(element, temp_in, endA - startA);
        temp_out[k] = element;
    }
    __syncthreads();

    // salva output
    output = output + 2 * couple_block_id * BLOCK_SIZE;
    for(int i = 0; i < (endA - startA) + (endB - startB); i++) {
        output[startA + startB + i] = temp_out[i];
    }
}

__global__
void uniform_merge3_kernel(int INPUT_ARRAY input, int * output, int INPUT_ARRAY indexA, int INPUT_ARRAY indexB, int INPUT_ARRAY a_in, int * a_out, float INPUT_ARRAY b_in, float * b_out, int len, int BLOCK_SIZE) {

    __shared__ int temp_in[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ int temp_out[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ int temp_a_in[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ int temp_a_out[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ float temp_b_in[2 * SEGMERGE_SM_SPLITTER_DISTANCE];
    __shared__ float temp_b_out[2 * SEGMERGE_SM_SPLITTER_DISTANCE];

    for(int i = 0; i < 2 * SEGMERGE_SM_SPLITTER_DISTANCE; i++) {
        temp_in[i] = 0;
        temp_out[i] = 0;
        temp_a_in[i] = 0;
        temp_a_out[i] = 0;
        temp_b_in[i] = 0;
        temp_b_out[i] = 0;
    }

    // processa l'elemento dello splitter
    const int splitter_index = blockIdx.x;

    // recupera blocco sul quale stai lavorando
    const int SPLITTER_PER_BLOCK = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    int couple_block_id = splitter_index / (2 * SPLITTER_PER_BLOCK);
    int item = splitter_index % (2 * SPLITTER_PER_BLOCK);

    // recupera estremi sui quali lavorare
    int startA = (item == 0) ? 0 : indexA[splitter_index-1];
    int startB = (item == 0) ? 0 : indexB[splitter_index-1];
    int endA   = indexA[splitter_index];
    int endB   = indexB[splitter_index];    
    if(endA - startA > SEGMERGE_SM_SPLITTER_DISTANCE) printf("!!!Error A[%d:%d] > %d SM SP S\n", startA, endA, SEGMERGE_SM_SPLITTER_DISTANCE);
    if(endB - startB > SEGMERGE_SM_SPLITTER_DISTANCE) printf("!!!Error B[%d:%d] > %d SM SP S\n", startB, endB, SEGMERGE_SM_SPLITTER_DISTANCE);
    
    // carico gli elementi in temp_in
    int OFFSET_A = 2*couple_block_id*BLOCK_SIZE;
    int OFFSET_B = (2*couple_block_id+1)*BLOCK_SIZE;
    utils::cuda::devcopy<int>(temp_in, 
        input + OFFSET_A + startA, endA - startA);
    utils::cuda::devcopy<int>(temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, 
        input + OFFSET_B + startB, endB - startB);

    utils::cuda::devcopy<int>(temp_a_in, 
        a_in  + OFFSET_A + startA, endA - startA);
    utils::cuda::devcopy<int>(temp_a_in + SEGMERGE_SM_SPLITTER_DISTANCE, 
        a_in  + OFFSET_B + startB, endB - startB);

    utils::cuda::devcopy<float>(temp_b_in, 
        b_in  + OFFSET_A + startA, endA - startA);
    utils::cuda::devcopy<float>(temp_b_in + SEGMERGE_SM_SPLITTER_DISTANCE, 
        b_in  + OFFSET_B + startB, endB - startB);

    //printf("(%2d): COUPLE=%2d, OFFSET_A=%2d, OFFSET_B=%2d, A[%2d:%2d] B[%2d:%2d]\n"
    //       "temp__ = [%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d]\n"
    //       "temp_a = [%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d]\n"
    //       "temp_b = [%2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f]\n",
    //    splitter_index,
    //    couple_block_id, OFFSET_A, OFFSET_B,
    //    startA, endA, startB, endB,
    //    temp_in[0], temp_in[1], temp_in[2], temp_in[3],
    //    temp_in[4], temp_in[5], temp_in[6], temp_in[7],
    //    temp_a_in[0], temp_a_in[1], temp_a_in[2], temp_a_in[3],
    //    temp_a_in[4], temp_a_in[5], temp_a_in[6], temp_a_in[7],
    //    temp_b_in[0], temp_b_in[1], temp_b_in[2], temp_b_in[3],
    //    temp_b_in[4], temp_b_in[5], temp_b_in[6], temp_b_in[7]
    //);

    // effettuo merge
    for(int i = 0; i < endA - startA; i++) {
        int element = temp_in[i];
        int elementA = temp_a_in[i];
        int elementB = temp_b_in[i];
        int posInA = i;
        int posInB = find_position_in_sorted_array(element, temp_in + SEGMERGE_SM_SPLITTER_DISTANCE, endB - startB);
        int k = posInA + posInB;
        temp_out[k] = element;
        temp_a_out[k] = elementA;
        temp_b_out[k] = elementB;
    }
    __syncthreads();

    for(int i = 0; i < endB - startB; i++) {
        int element = temp_in[SEGMERGE_SM_SPLITTER_DISTANCE + i];
        int elementA = temp_a_in[SEGMERGE_SM_SPLITTER_DISTANCE + i];
        int elementB = temp_b_in[SEGMERGE_SM_SPLITTER_DISTANCE + i];
        int k = i + find_last_position_in_sorted_array(element, temp_in, endA - startA);
        temp_out[k] = element;
        temp_a_out[k] = elementA;
        temp_b_out[k] = elementB;
    }
    __syncthreads();

    //printf("OUT\n");
    //printf("(%2d): COUPLE=%2d, OFFSET_A=%2d, OFFSET_B=%2d, A[%2d:%2d] B[%2d:%2d]\n"
    //       "temp__ = [%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d]\n"
    //       "temp_a = [%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d]\n"
    //       "temp_b = [%2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f, %2.0f]\n",
    //    splitter_index,
    //    couple_block_id, OFFSET_A, OFFSET_B,
    //    startA, endA, startB, endB,
    //    temp_out[0], temp_out[1], temp_out[2], temp_out[3],
    //    temp_out[4], temp_out[5], temp_out[6], temp_out[7],
    //    temp_a_out[0], temp_a_out[1], temp_a_out[2], temp_a_out[3],
    //    temp_a_out[4], temp_a_out[5], temp_a_out[6], temp_a_out[7],
    //    temp_b_out[0], temp_b_out[1], temp_b_out[2], temp_b_out[3],
    //    temp_b_out[4], temp_b_out[5], temp_b_out[6], temp_b_out[7]
    //);

    // salva output
    for(int i = 0; i < (endA - startA) + (endB - startB); i++) {
        output[OFFSET_A + startA + startB + i] = temp_out[i];
        a_out [OFFSET_A + startA + startB + i] = temp_a_out[i];
        b_out [OFFSET_A + startA + startB + i] = temp_b_out[i];
    }
}

void procedures::cuda::segmerge_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {
    
    DPRINT_MSG("\n\n\n##### Starting segmerge_sm_step")
    DPRINT_ARR_CUDA(input, len)

    // 1. lavoro su coppie di blocchi per estrarre gli splitter e gli indici necessari a lavorarci sopra
    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    const int COUPLE_OF_BLOCKS = BLOCK_NUMBER / 2;
    const int SPLITTER_PER_BLOCKS = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    const int SPLITTER_PER_LAST_BLOCK = (len % BLOCK_SIZE == 0) ? SPLITTER_PER_BLOCKS : DIV_THEN_CEIL(len % BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    const int SPLITTER_NUMBER = 
        (BLOCK_NUMBER%2==1)                      ? // il numero di blocchi da processare è dispari?
        (2*COUPLE_OF_BLOCKS*SPLITTER_PER_BLOCKS) : // se si, tutti i blocchi sa processare hanno dimensione piena
        (2*(COUPLE_OF_BLOCKS-1)*SPLITTER_PER_BLOCKS+SPLITTER_PER_BLOCKS+SPLITTER_PER_LAST_BLOCK); // se no, l'ultimo blocco da processare potrebbe avere lunghezza minore

    //printf("SPLITTER_NUMBER=%d, SPLITTER_PER_BLOCK=%d, SPLITTER_PER_LAST_BLOCK=%d, BLOCK_NUMBER=%d, COUPLE_OF_BLOCKS=%d\n", 
    //    SPLITTER_NUMBER, SPLITTER_PER_BLOCKS, SPLITTER_PER_LAST_BLOCK, BLOCK_NUMBER, COUPLE_OF_BLOCKS);

    int * splitter     = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * splitter_out = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);

    splitter_kernel<<<COUPLE_OF_BLOCKS, 1024>>>(input, splitter, indexA, indexB, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR
    DPRINT_MSG("After splitter_kernel")
    DPRINT_ARR_CUDA(splitter, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexA, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexB, SPLITTER_NUMBER)

    // 2. riordino per blocchi l'array degli splitter e gli indici ad esso associati
    segmerge3_step(splitter, splitter_out, SPLITTER_NUMBER, SPLITTER_PER_BLOCKS, indexA, indexA_out, indexB, indexB_out);
    DPRINT_MSG("After segmerge3_sm_step")
    DPRINT_ARR_CUDA(splitter_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexA_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexB_out, SPLITTER_NUMBER)

    // 3. sistemo gli indici di `indexA`, `indexB` per evitare merge vuoti
    fix_indexes_kernel<<<COUPLE_OF_BLOCKS, 1>>>(indexA_out, indexB_out, len, BLOCK_SIZE, SPLITTER_NUMBER, SPLITTER_PER_BLOCKS);
    CUDA_CHECK_ERROR
    DPRINT_MSG("After fix_indexes_kernel")
    DPRINT_ARR_CUDA(splitter_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexA_out, SPLITTER_NUMBER)
    DPRINT_ARR_CUDA(indexB_out, SPLITTER_NUMBER)

    // 4. eseguo il merge di porzioni di blocchi di dimensione uniforme
    uniform_merge_kernel<<<SPLITTER_NUMBER, 1>>>(input, output, indexA_out, indexB_out, len, BLOCK_SIZE);
    CUDA_CHECK_ERROR

    // 5. eventualmente copio il risultato dell' ultimo blocco di array rimasto spaiato
    if(BLOCK_NUMBER % 2 == 1) {
        utils::cuda::copy<int>(
            output + 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            input +  2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            len - 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE
        );
    }

    DPRINT_MSG("After uniform_merge_kernel")
    DPRINT_ARR_CUDA(input, len)
    DPRINT_ARR_CUDA(output, len)

    utils::cuda::deallocate(splitter);
    utils::cuda::deallocate(indexA);
    utils::cuda::deallocate(indexB);
    utils::cuda::deallocate(splitter_out);
    utils::cuda::deallocate(indexA_out);
    utils::cuda::deallocate(indexB_out);
}

void procedures::cuda::segmerge3_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, float INPUT_ARRAY b_in, float * b_out) {
    
    DPRINT_MSG("\n\n\n##### Starting segmerge3_sm_step")
    DPRINT_ARR_CUDA(input, len)

    CUDA_CHECK_ERROR

    // 1. lavoro su coppie di blocchi per estrarre gli splitter e gli indici necessari a lavorarci sopra
    const int BLOCK_NUMBER = DIV_THEN_CEIL(len, BLOCK_SIZE);
    const int COUPLE_OF_BLOCKS = BLOCK_NUMBER / 2;
    const int SPLITTER_PER_BLOCKS = DIV_THEN_CEIL(BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    const int SPLITTER_PER_LAST_BLOCK = (len % BLOCK_SIZE == 0) ? SPLITTER_PER_BLOCKS : DIV_THEN_CEIL(len % BLOCK_SIZE, SEGMERGE_SM_SPLITTER_DISTANCE);
    const int SPLITTER_NUMBER = 
        (BLOCK_NUMBER%2==1)                      ? // il numero di blocchi da processare è dispari?
        (2*COUPLE_OF_BLOCKS*SPLITTER_PER_BLOCKS) : // se si, tutti i blocchi sa processare hanno dimensione piena
        (2*(COUPLE_OF_BLOCKS-1)*SPLITTER_PER_BLOCKS+SPLITTER_PER_BLOCKS+SPLITTER_PER_LAST_BLOCK); // se no, l'ultimo blocco da processare potrebbe avere lunghezza minore

    //printf("SPLITTER_NUMBER=%d, SPLITTER_PER_BLOCK=%d, SPLITTER_PER_LAST_BLOCK=%d, BLOCK_NUMBER=%d, COUPLE_OF_BLOCKS=%d\n", 
    //    SPLITTER_NUMBER, SPLITTER_PER_BLOCKS, SPLITTER_PER_LAST_BLOCK, BLOCK_NUMBER, COUPLE_OF_BLOCKS);

    int * splitter     = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB       = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * splitter_out = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexA_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);
    int * indexB_out   = utils::cuda::allocate<int>(SPLITTER_NUMBER);

    if(COUPLE_OF_BLOCKS > 0) {
        
        splitter_kernel<<<COUPLE_OF_BLOCKS, 1024>>>(input, splitter, indexA, indexB, len, BLOCK_SIZE);
        CUDA_CHECK_ERROR
        DPRINT_MSG("After splitter_kernel")
        DPRINT_ARR_CUDA(splitter, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexA, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexB, SPLITTER_NUMBER)

        // 2. riordino per blocchi l'array degli splitter e gli indici ad esso associati
        segmerge3_step(splitter, splitter_out, SPLITTER_NUMBER, SPLITTER_PER_BLOCKS, indexA, indexA_out, indexB, indexB_out);
        DPRINT_MSG("After segmerge3_sm_step")
        DPRINT_ARR_CUDA(splitter_out, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexA_out, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexB_out, SPLITTER_NUMBER)

        // 3. sistemo gli indici di `indexA`, `indexB` per evitare merge vuoti
        fix_indexes_kernel<<<COUPLE_OF_BLOCKS, 1>>>(indexA_out, indexB_out, len, BLOCK_SIZE, SPLITTER_NUMBER, SPLITTER_PER_BLOCKS);
        CUDA_CHECK_ERROR
        DPRINT_MSG("After fix_indexes_kernel")
        DPRINT_ARR_CUDA(splitter_out, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexA_out, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(indexB_out, SPLITTER_NUMBER)
        DPRINT_ARR_CUDA(a_in, len)
        DPRINT_ARR_CUDA(b_in, len)

        // 4. eseguo il merge di porzioni di blocchi di dimensione uniforme
        uniform_merge3_kernel<<<SPLITTER_NUMBER, 1>>>(input, output, indexA_out, indexB_out, a_in, a_out, b_in, b_out, len, BLOCK_SIZE);
        CUDA_CHECK_ERROR
    }

    // 5. eventualmente copio il risultato dell' ultimo blocco di array rimasto spaiato
    if(BLOCK_NUMBER % 2 == 1) {
        utils::cuda::copy<int>(
            output + 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            input +  2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            len - 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE
        );
        utils::cuda::copy<int>(
            a_out + 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            a_in +  2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            len - 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE
        );
        utils::cuda::copy<float>(
            b_out + 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            b_in +  2 * COUPLE_OF_BLOCKS * BLOCK_SIZE, 
            len - 2 * COUPLE_OF_BLOCKS * BLOCK_SIZE
        );
    }

    DPRINT_MSG("After uniform_merge_kernel")
    DPRINT_ARR_CUDA(input, len)
    DPRINT_ARR_CUDA(output, len)

    utils::cuda::deallocate(splitter);
    utils::cuda::deallocate(indexA);
    utils::cuda::deallocate(indexB);
    utils::cuda::deallocate(splitter_out);
    utils::cuda::deallocate(indexA_out);
    utils::cuda::deallocate(indexB_out);

}

void procedures::reference::segmerge_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE) {
    segmerge_step(input, output, len, BLOCK_SIZE);
}

void procedures::reference::segmerge3_sm_step(int INPUT_ARRAY input, int * output, int len, int BLOCK_SIZE, int INPUT_ARRAY a_in, int * a_out, float INPUT_ARRAY b_in, float * b_out) {
    segmerge3_step(input, output, len, BLOCK_SIZE, a_in, a_out, b_in, b_out);
}