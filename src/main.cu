#include <iostream>
#include <iomanip>
#include "hip/hip_runtime_api.h"
#include "libfort/fort.hpp"
#include "tester/Tester.hh"
#include "transposers/ScanTransposer.hh"
#include "transposers/CusparseTransposer.hh"

#define REPETITION_NUMBER 100

int main(int argc, char **argv) {

    findCudaDevice(argc, (const char **) argv);

    CusparseTransposer cu;
    ScanTransposer sc;
    ScanTransposer sc2(256, 1024);
    ScanTransposer sc3(256, 1536);
    Tester tester;
    tester.add_test(   10,    10,       20, REPETITION_NUMBER);
    tester.add_test(  100,   100,     1000, REPETITION_NUMBER);
    tester.add_test( 1000,  1000,    10000, REPETITION_NUMBER);
    tester.add_test(10000, 10000,  1000000, REPETITION_NUMBER);
    tester.add_test(10000, 10000, 10000000, REPETITION_NUMBER);
    tester.add_processor(&cu, "CUSPARSE");
    tester.add_processor(&sc, "SCAN256-256");
    tester.add_processor(&sc2, "SCAN256-1k");
    tester.add_processor(&sc3, "SCAN256-1.5k");
    tester.run();
    tester.print();

    // ScanTransposer sc(256, 5);
    // Tester tester;
    // tester.add_test(5, 4, 10, REPETITION_NUMBER);
    // tester.add_processor(&sc, "SCANTRANS");
    // tester.run(true);
    // tester.print();
    
    return 0;
}