#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "procedures.hh"

void procedures::reference::scan(int INPUT_ARRAY input, int * output, int len) {
    output[0] = 0;
    for(int i = 1; i < len; i++) {
        output[i] = output[i-1] + input[i-1];
    }
}

static const int THREADS_PER_BLOCK = 512;

static const int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

void scan_small(int * output, int INPUT_ARRAY input, int length);

void scan_even(int * output, int INPUT_ARRAY input, int length);

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo);

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums);

void add_multiple_offset(int * output, int BLOCKS, int INPUT_ARRAY incr);

void add_single_offset(int * output, int length, int INPUT_ARRAY n1, int INPUT_ARRAY n2);


void procedures::cuda::scan(int INPUT_ARRAY d_in, int * d_out, int length) {

	if (length <= ELEMENTS_PER_BLOCK) {

		scan_small(d_out, d_in, length);
	}
	else if(length % ELEMENTS_PER_BLOCK == 0) {

		scan_even(d_out, d_in, length);
	} else {

		// perform a large scan on a compatible multiple of elements
		int remainder = length % ELEMENTS_PER_BLOCK;
		int lengthMultiple = length - remainder;
		scan_even(d_out, d_in, lengthMultiple);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		scan_small(startOfOutputArray, &(d_in[lengthMultiple]), remainder);

		add_single_offset(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}

	return;
}

void scan_small(int * output, int INPUT_ARRAY input, int length) {
	// non alloco tutta la memoria massima ma solo quella che mi serve
	int LENGHT_TWO_POW = utils::next_two_pow(length);

	// chiamo il kernel che non necessita di somme
    prescan_arbitrary_unoptimized<<<1, DIV_THEN_CEIL(length, 2), 2 * LENGHT_TWO_POW * sizeof(int)>>>(
		output, input, length, LENGHT_TWO_POW);
}

void scan_even(int * output, int INPUT_ARRAY input, int length) {

	// quanti blocchi? la lunghezza è multipla di ELEMENTS_PER_BLOCK
	const int BLOCKS = length / ELEMENTS_PER_BLOCK;
	const int SM_SIZE = 2 * ELEMENTS_PER_BLOCK * sizeof(int);

	// alloco array ausiliari degli offset
	int * sums = utils::cuda::allocate<int>(BLOCKS);
	int * incr = utils::cuda::allocate<int>(BLOCKS);

	// chiamo il kernel (mantengo gli offset parziali in sums)
	prescan_large_unoptimized<<<BLOCKS, THREADS_PER_BLOCK, SM_SIZE>>>(
		output, input, ELEMENTS_PER_BLOCK, sums);

	// applico prefix-scan sugli offset parziali
	procedures::cuda::scan(sums, incr, BLOCKS);

	// sommo gli offset ad output
	add_multiple_offset(output, BLOCKS, incr);

	utils::cuda::deallocate(sums);
	utils::cuda::deallocate(incr);
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	if (threadID < n) {
		temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
		temp[2 * threadID + 1] = input[2 * threadID + 1];
	}
	else { 
		temp[2 * threadID] = 0;
		temp[2 * threadID + 1] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[2 * threadID] = temp[2 * threadID]; // write results to device memory
		output[2 * threadID + 1] = temp[2 * threadID + 1];
	}
}


__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}

// ==========================================================================
// ADD UTILITY ==============================================================
//==========================================================================

__global__ 
void add(int *output, int INPUT_ARRAY n1, int INPUT_ARRAY n2) {
	int i = threadIdx.x;
	output[i] += n1[0] + n2[0];
}

__global__ 
void add(int * output, int * incr) {
	int j = blockIdx.x;
	int i = threadIdx.x;
	output[j*ELEMENTS_PER_BLOCK + i] += incr[j];
}

void add_single_offset(int * output, int length, int INPUT_ARRAY n1, int INPUT_ARRAY n2) {
	add<<<1, length>>>(output, n1, n2);
}

void add_multiple_offset(int * output, int BLOCKS, int INPUT_ARRAY incr) {
	add<<<BLOCKS, ELEMENTS_PER_BLOCK>>>(output, incr);
}
