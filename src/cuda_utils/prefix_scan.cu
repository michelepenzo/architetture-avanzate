#include "hip/hip_runtime.h"
#include "cuda_utils/prefix_scan.hh"
#include "transposers/CudaTransposer.hh"
#include <iostream>
#include <iomanip>

int THREADS_PER_BLOCK = 8;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

long sequential_scan(int* output, int* input, int length) {
	long start_time = get_nanos();

	output[0] = 0; // since this is a prescan, not a scan
	for (int j = 1; j < length; ++j)
	{
		output[j] = input[j - 1] + output[j - 1];
	}

	long end_time = get_nanos();
	return end_time - start_time;
}

float blockscan(int *output, int *input, int length, bool bcao) {
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int powerOfTwo = nextPowerOfTwo(length);
	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

float scan(int *output, int *input, int length, bool bcao) {
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_out, d_in, length, bcao);
	}

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

void scan_on_cuda(int *d_out, int *d_in, int length, bool bcao) {

	int *d_in_host  = new int[length];
	int *d_out_host = new int[length];

	SAFE_CALL(hipMemcpy( d_in_host,  d_in, length*sizeof(int), hipMemcpyDeviceToHost));
	std::cout << "\n\n### scan_on_cuda: d_in_host  : ";
	for(int i = 0; i < length; i++) {
        std::cout << std::setw(2) << d_in_host[i] << " ";
    }
	std::cout << " (" << d_in_host[length] << ")" << "\n";
	
	if (length > ELEMENTS_PER_BLOCK) {
		std::cout << "scan_on_cuda: length=" << length << " chooses scanLargeDeviceArray\n";
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		std::cout << "scan_on_cuda: length=" << length << " chooses scanSmallDeviceArray\n";
		scanSmallDeviceArray(d_out, d_in, length, bcao);
		std::cout << "scan_on_cuda: ended scanSmallDeviceArray" << std::endl;
	}

	SAFE_CALL(hipMemcpy( d_in_host,  d_in, length*sizeof(int), hipMemcpyDeviceToHost));
	std::cout << "\n\n### scan_on_cuda: d_in_host  : ";
	for(int i = 0; i < length; i++) {
        std::cout << std::setw(2) << d_in_host[i] << " ";
    }
	std::cout << " (" << d_in_host[length] << ")" << "\n";

	SAFE_CALL(hipMemcpy(d_out_host, d_out, length*sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "### scan_on_cuda: d_out_host : ";
	for(int i = 0; i < length; i++) {
        std::cout << std::setw(2) << d_out_host[i] << " ";
    }
	std::cout << " (" << d_out_host[length] << ")" << "\n";

	delete d_in_host;
	delete d_out_host;
	
}

__global__ void my_add(int *array, int len) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < len) {

		const int OFFSET = *(array - 1);

		array[i] = array[i] + OFFSET;
	}
}

void scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	std::cout << "scanLargeDeviceArray: remainder " << remainder << "\n";
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		std::cout << "scanLargeDeviceArray: **** 16 elem ARRAY **** from=0 to=" << lengthMultiple << "\n";
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		std::cout << "scanLargeDeviceArray: **** 1 elem ARRAY **** from=" << lengthMultiple << " len=" << remainder << "\n";
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

		std::cout << "scanLargeDeviceArray: add\n";

		//const int OFFSET = d_out[lengthMultiple-1]; // ultimo elemento processato da `scanLargeEvenDeviceArray`

		// a tutti gli elementi di d_out[lengthMultiple...length] aggiungo OFFSET
		//for(int j = lengthMultiple; j < length; j++) {
		//	d_out[j] += OFFSET;
		//}

		my_add<<<1, remainder>>>(startOfOutputArray, remainder);

		//add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	
		
		/*__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}*/
	
	
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	
	int powerOfTwo = nextPowerOfTwo(length);

	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	
	/*std::cout << "\t scanSmallDeviceArray: length=" << length << "; powerOfTwo=" << powerOfTwo << "\n";

	int *d_out_host = new int[length];
	int *d_in_host  = new int[length];
	//SAFE_CALL(hipMemcpy(d_out_host, d_out, length*sizeof(int), hipMemcpyDeviceToHost));
    SAFE_CALL(hipMemcpy( d_in_host,  d_in, length*sizeof(int), hipMemcpyDeviceToHost));
	
	std::cout << "d_in_host : ";
	for(int i = 0; i < length; i++) {
        std::cout << std::setw(2) << d_in_host[i] << " ";
    }
	std::cout << "\n";
	
	int n = length;	


	// copio `d_in_host` dentro `d_out_host`
	for(int i = 0; i < length; i++) {
		const int temp = d_in_host[i];
        d_out_host[i] = temp;
    }

	// applico prefix sum "in-place" che so che funziona (?)
	for(int i = 0; i < n; i++) {
        d_out_host[i+1] += d_out_host[i];
    }
    for(int i = n-1; i >= 0; i--) {
        d_out_host[i+1] = d_out_host[i];
    }
    d_out_host[0] = 0;



	std::cout << "d_out_host: ";
	for(int i = 0; i < length; i++) {
        std::cout << std::setw(2) << d_out_host[i] << " ";
    }
	std::cout << "\n";

	SAFE_CALL(hipMemcpy(d_out, d_out_host, length*sizeof(int), hipMemcpyHostToDevice));
    //SAFE_CALL(hipMemcpy( d_in,  d_in_host, length*sizeof(int), hipMemcpyHostToDevice));
    
	delete d_out_host;
	delete d_in_host; */
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length, bool bcao) {

	std::cout << "\t scanLargeEvenDeviceArray: length " << length << "\n";
	
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	if (bcao) {
		prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}
	else {
		prescan_large_unoptimized<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// perform a large scan on the sums arr
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// only need one block to scan sums arr so can use small scan
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5

// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo)
{
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	if (threadID < n) {
		temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
		temp[2 * threadID + 1] = input[2 * threadID + 1];
	}
	else {
		temp[2 * threadID] = 0;
		temp[2 * threadID + 1] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[2 * threadID] = temp[2 * threadID]; // write results to device memory
		output[2 * threadID + 1] = temp[2 * threadID + 1];
	}
}


__global__ void prescan_large(int *output, int *input, int n, int *sums) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}


__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

// from https://stackoverflow.com/a/3638454
bool isPowerOfTwo(int x) {
	return x && !(x & (x - 1));
}

// from https://stackoverflow.com/a/12506181
int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}


// from https://stackoverflow.com/a/36095407
// Get the current time in nanoseconds
long get_nanos() {
	struct timespec ts;
	timespec_get(&ts, TIME_UTC);
	return (long)ts.tv_sec * 1000000000L + ts.tv_nsec;
}
