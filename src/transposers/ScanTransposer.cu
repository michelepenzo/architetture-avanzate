#include "hip/hip_runtime.h"
#include "transposers/ScanTransposer.hh"

__global__ 
void scan_trans_kernel(
    int m, int n, int nnz, 
    int *csrRowPtr, int *csrColIdx, float *csrVal,
    int *cscColPtr, int *cscRowIdx, float *cscVal, 
    int *inter, int *intra)
{

    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // intra e inter
    if (global_id < nnz)
    {
        int index = (global_id + 1) * n + csrColIdx[global_id];
        intra[global_id] = inter[index];
        inter[index] = inter[index] + 1;
    }
}

int ScanTransposer::csr2csc_gpumemory(int m, int n, int nnz, int *csrRowPtr, int *csrColIdx, float *csrVal, int *cscColPtr, int *cscRowIdx, float *cscVal)
{

    const int N = nnz;
    const int BLOCK_SIZE_X = 256;
    int *intra, *inter, *intra_host, *inter_host;

    // resource allocation
    intra_host = new int[nnz]();
    inter_host = new int[(N + 1) * n]();
    hipMalloc(&intra, (nnz) * sizeof(int));
    hipMalloc(&inter, ((N + 1) * n) * sizeof(int));

    // pass info
    SAFE_CALL(hipMemcpy(intra, intra_host, (nnz) * sizeof(int),         hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(inter, inter_host, ((N + 1) * n) * sizeof(int), hipMemcpyHostToDevice));

    // kernel execution
    dim3 DimGrid(N / BLOCK_SIZE_X, 1, 1);
    if (N % BLOCK_SIZE_X)
        DimGrid.x++;
    dim3 DimBlock(BLOCK_SIZE_X, 1, 1);

    scan_trans_kernel<<<DimGrid, DimBlock>>>(
        m, n, nnz,
        csrRowPtr, csrColIdx, csrVal,
        cscColPtr, cscRowIdx, cscVal,
        inter, intra);
    CHECK_CUDA_ERROR

    // retrieve info
    SAFE_CALL(hipMemcpy(intra_host, intra, (nnz) * sizeof(int),         hipMemcpyDeviceToHost));
    SAFE_CALL(hipMemcpy(inter_host, inter, ((N + 1) * n) * sizeof(int), hipMemcpyDeviceToHost));

    // debug prints
    std::cout << "Intra: " << std::endl;
    for (int i = 0; i < nnz; i++)
    {
        std::cout << intra_host[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Inter: ";
    for (int i = 0; i < (N + 1); i++)
    {
        std::cout << std::endl << "Row " << i - 1 << ": ";
        for (int j = 0; j < n; j++)
        {
            std::cout << inter_host[i * n + j] << " ";
        }
    }
    std::cout << std::endl;

    // resource deallocation
    delete[] intra_host;
    delete[] inter_host;
    hipFree(intra);
    hipFree(inter);

    return COMPUTATION_ERROR;
}